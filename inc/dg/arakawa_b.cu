#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "backend/timer.cuh"
#include "arakawa.h"
#include "blas.h"



const double lx = 2*M_PI;
const double ly = 2*M_PI;
//const double lx = 1.;
//const double ly = 1.;


//choose some mean function (attention on lx and ly)
//THESE ARE NOT PERIODIC
/*
double left( double x, double y) { return sin(x)*cos(y);}
double right( double x, double y){ return exp(0.1*(x+y)); }
double jacobian( double x, double y) 
{
    return exp( x-M_PI)*(sin(x)+cos(x))*sin(y) * exp(y-M_PI)*sin(x)*(sin(y) + cos(y)) - sin(x)*exp(x-M_PI)*cos(y) * cos(x)*sin(y)*exp(y-M_PI); 
}
*/

dg::bc bcx = dg::PER;
dg::bc bcy = dg::PER;
double left( double x, double y) {return sin(x)*cos(y);}
double right( double x, double y) {return cos(x)*sin(y);}
double jacobian( double x, double y) 
{
    return cos(x)*cos(y)*cos(x)*cos(y) - sin(x)*sin(y)*sin(x)*sin(y); 
}
////These are for comparing to FD arakawa results
//double left( double x, double y) {return sin(2.*M_PI*(x-hx/2.));}
//double right( double x, double y) {return y;}
//double jacobian( double x, double y) {return 2.*M_PI*cos(2.*M_PI*(x-hx/2.));}
typedef dg::DVec Vector;
typedef dg::DMatrix Matrix;

int main(int argc, char* argv[])
{
    std::cout << std::fixed<<"\nTEST 2D VERSION!!\n";
    dg::Timer t;
    unsigned n, Nx, Ny;
    if(argc != 4) {
        std::cout << "ERROR: exactly three arguments n, Nx, Ny are required" << std::endl;
        exit(1);
    }
    n = atoi(argv[1]); Nx = atoi(argv[2]); Ny = atoi(argv[3]);
    dg::Grid2d grid( 0, lx, 0, ly, n, Nx, Ny, dg::PER, dg::PER);
    Vector w2d = dg::create::weights( grid);
    std::cout << "Computing on the Grid " <<n<<" x "<<Nx<<" x "<<Ny <<std::endl;
    Vector lhs = dg::evaluate ( left, grid), jac(lhs);
    Vector rhs = dg::evaluate ( right,grid);
    const Vector sol = dg::evaluate( jacobian, grid );
    Vector eins = dg::evaluate( dg::one, grid );
    //std::cout<< std::setprecision(2);

    dg::ArakawaX<dg::CartesianGrid2d, Matrix, Vector> arakawa( grid);
    unsigned multi=20;
    t.tic(); 
    for( unsigned i=0; i<multi; i++)
        arakawa( lhs, rhs, jac);
    t.toc();
    std::cout << "Arakawa took "<<t.diff()*1000/(double)multi<<"ms\n";

    std::cout << std::scientific;
    std::cout << "Mean     Jacobian is "<<dg::blas2::dot( eins, w2d, jac)<<"\n";
    std::cout << "Mean rhs*Jacobian is "<<dg::blas2::dot( rhs, w2d, jac)<<"\n";
    std::cout << "Mean   n*Jacobian is "<<dg::blas2::dot( lhs, w2d, jac)<<"\n";
    dg::blas1::axpby( 1., sol, -1., jac);
    std::cout << "Distance to solution "<<sqrt(dg::blas2::dot( w2d, jac))<<std::endl; //don't forget sqrt when comuting errors

    //periocid bc       |  dirichlet in x per in y
    //n = 1 -> p = 2    |        1.5
    //n = 2 -> p = 1    |        1
    //n = 3 -> p = 3    |        3
    //n = 4 -> p = 3    |        3
    //n = 5 -> p = 5    |        5
    // quantities are all conserved to 1e-15 for periodic bc
    // for dirichlet bc these are not better conserved than normal jacobian

    return 0;
}
