#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>

#include <limits.h>
#include <unistd.h>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <mpi.h>

#include "backend/timer.cuh"

#include "arakawa.h"
#include "backend/mpi_init.h"
#include "backend/typedefs.cuh"



const double lx = 2*M_PI;
const double ly = 2*M_PI;
//const double lx = 1.;
//const double ly = 1.;


//choose some mean function (attention on lx and ly)
//THESE ARE NOT PERIODIC
/*
double left( double x, double y) { return sin(x)*cos(y);}
double right( double x, double y){ return exp(0.1*(x+y)); }
double jacobian( double x, double y) 
{
    return exp( x-M_PI)*(sin(x)+cos(x))*sin(y) * exp(y-M_PI)*sin(x)*(sin(y) + cos(y)) - sin(x)*exp(x-M_PI)*cos(y) * cos(x)*sin(y)*exp(y-M_PI); 
}
*/

dg::bc bcx = dg::PER;
dg::bc bcy = dg::PER;
double left( double x, double y) {return sin(x)*cos(y);}
double right( double x, double y) {return cos(x)*sin(y);}
double jacobian( double x, double y) 
{
    return cos(x)*cos(y)*cos(x)*cos(y) - sin(x)*sin(y)*sin(x)*sin(y); 
}
////These are for comparing to FD arakawa results
//double left( double x, double y) {return sin(2.*M_PI*(x-hx/2.));}
//double right( double x, double y) {return y;}
//double jacobian( double x, double y) {return 2.*M_PI*cos(2.*M_PI*(x-hx/2.));}


void mpi_init2d_cli( dg::bc bcx, dg::bc bcy, unsigned& n, unsigned& Nx, unsigned& Ny, MPI_Comm& comm, char argc, char* argv[]  )
{
    int periods[2] = {false,false};
    if( bcx == dg::PER) periods[0] = true;
    if( bcy == dg::PER) periods[1] = true;
    int rank, size;
    MPI_Comm_rank( MPI_COMM_WORLD, &rank);
    MPI_Comm_size( MPI_COMM_WORLD, &size);
    std::cout << rank << " " << size << std::endl;

    if(argc != 6 && argc != 7) {
        std::cerr << "ERROR: 5 or 6 arguments are required: n, Nx, Ny, npx, npy, [gpus_per_node]" << std::endl;
        exit(1);
    }
#ifdef __NVCC__
    if(argc == 7)
        hipSetDevice(rank % atoi(argv[6]));
#endif

    //exit(1);
    if(rank==0)std::cout << "MPI v"<<MPI_VERSION<<"."<<MPI_SUBVERSION<<std::endl;
    int np[2];
    np[0] = atoi(argv[4]); np[1] = atoi(argv[5]);
    if( rank == 0)
    {
        std::cout<< "Computing with "<<np[0] <<" x "<<np[1]<<" = "<<size<<" processes! "<<std::endl;
        assert( size == np[0]*np[1]);
    }
    MPI_Bcast( np, 2, MPI_INT, 0, MPI_COMM_WORLD);
    MPI_Cart_create( MPI_COMM_WORLD, 2, np, periods, true, &comm);
    n = atoi(argv[1]); Nx = atoi(argv[2]); Ny = atoi(argv[3]);
    if( rank == 0)
    {
        std::cout<< "On the grid "<<n <<" x "<<Nx<<" x "<<Ny<<std::endl;
    }
    MPI_Bcast(  &n,1 , MPI_UNSIGNED, 0, comm);
    MPI_Bcast( &Nx,1 , MPI_UNSIGNED, 0, comm);
    MPI_Bcast( &Ny,1 , MPI_UNSIGNED, 0, comm);
}


int main(int argc, char* argv[])
{
    MPI_Init( &argc, &argv);
    int rank;

    char host[100];
    gethostname(host,100);

    int size;
    MPI_Comm_rank( MPI_COMM_WORLD, &rank);
    MPI_Comm_size( MPI_COMM_WORLD, &size);
    std::cout << rank << " " << size << " " << host << std::endl;

    unsigned n, Nx, Ny; 
    MPI_Comm comm;
    mpi_init2d_cli( bcx, bcy, n, Nx, Ny, comm, argc, argv);
    dg::MPIGrid2d grid( 0, lx, 0, ly, n, Nx, Ny, bcx, bcy, comm);
    MPI_Comm_rank( MPI_COMM_WORLD, &rank);
    dg::Timer t;
    dg::MDVec w2d = dg::create::weights( grid);
    dg::MDVec lhs = dg::evaluate ( left, grid), jac(lhs);
    dg::MDVec rhs = dg::evaluate ( right,grid);
    const dg::MDVec sol = dg::evaluate( jacobian, grid );
    dg::MDVec eins = dg::evaluate( dg::one, grid );
    std::cout<< std::setprecision(3);

    dg::ArakawaX<dg::CartesianMPIGrid2d, dg::MDMatrix, dg::MDVec> arakawa( grid);
    unsigned multi=20;
    t.tic(); 
    for( unsigned i=0; i<multi; i++)
        arakawa( lhs, rhs, jac);
    t.toc();
    if(rank==0)std::cout << "Arakawa took "<<t.diff()*1000/(double)multi<<"ms\n\n";

    double result = dg::blas2::dot( eins, w2d, jac);
    if(rank==0)std::cout << std::scientific;
    if(rank==0)std::cout << "Mean     Jacobian is "<<result<<"\n";
    result = dg::blas2::dot( rhs,  w2d, jac);
    if(rank==0)std::cout << "Mean rhs*Jacobian is "<<result<<"\n";
    result = dg::blas2::dot( lhs,  w2d, jac);
    if(rank==0)std::cout << "Mean lhs*Jacobian is "<<result<<"\n";
    dg::blas1::axpby( 1., sol, -1., jac);
    result = sqrt( dg::blas2::dot( w2d, jac));
    if(rank==0)std::cout << "Distance to solution "<<result<<std::endl; 


    MPI_Finalize();
    return 0;
}
