#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>

#include "cg.h"
#include "elliptic.h"

const double lx = 2.*M_PI;
const double ly = 2.*M_PI;

const double eps_ = 1e-6; //# of pcg iterations increases very much if 
 // eps << relativer Abstand der exakten Lösung zur Diskretisierung vom Sinus

double fct(double x, double y){ return sin(y)*sin(x);}
double laplace_fct( double x, double y) { return 2*sin(y)*sin(x);}
double initial( double x, double y) {return sin(0);}

int main()
{
    //global relative error in L2 norm is O(h^P)
    //more N means less iterations for same error
    unsigned n, Nx, Ny;
    std::cout << "Type n, Nx and Ny! \n";
    std::cin >> n >> Nx >> Ny;
    std::cout << "Computing on the Grid " <<n<<" x "<<Nx<<" x "<<Ny <<std::endl;
    dg::Grid2d grid( 0, lx, 0, ly,n, Nx, Ny, dg::PER, dg::PER);
    dg::HVec w2d = dg::create::weights( grid);
    dg::HVec v2d = dg::create::inv_weights( grid);
    std::cout<<"Evaluate initial condition\n";
    dg::HVec x = dg::evaluate( initial, grid);

    std::cout << "Create Laplacian\n";
    dg::Elliptic<dg::CartesianGrid2d, dg::HMatrix, dg::HVec> A( grid);
    dg::CG<dg::HVec > pcg( x, n*n*Nx*Ny);
    std::cout<<"Evaluate right hand side\n";
    dg::HVec b = dg::evaluate ( laplace_fct, grid);
    const dg::HVec solution = dg::evaluate ( fct, grid);
    //////////////////////////////////////////////////////////////////////
    //compute S b
    dg::blas2::symv( w2d, b, b);
    std::cout << "Number of pcg iterations "<< pcg( A, x, b, v2d, eps_)<<std::endl;
    //std::cout << "Number of cg iterations "<< pcg( A, x, b, dg::Identity<double>(), eps)<<std::endl;
    std::cout << "For a precision of "<< eps_<<std::endl;
    //compute error
    dg::HVec error( solution);
    dg::blas1::axpby( 1.,x,-1.,error);

    dg::HVec Ax(x), res( b);
    dg::blas2::symv(  A, x, Ax);
    dg::blas1::axpby( 1.,Ax,-1.,res);

    double xnorm = sqrt(dg::blas2::dot( w2d, x));
    std::cout << "L2 Norm of x0 is              " << xnorm << std::endl;
    double norm = sqrt(dg::blas2::dot(w2d , solution));
    std::cout << "L2 Norm of Solution is        " << norm << std::endl;
    double eps = sqrt(dg::blas2::dot(w2d , error));
    std::cout << "L2 Norm of Error is           " << eps << std::endl;
    double normres = sqrt(dg::blas2::dot( w2d, res));
    std::cout << "L2 Norm of Residuum is        " << normres << std::endl;
    std::cout << "L2 Norm of relative error is  " << eps/norm<<std::endl;
    //Fehler der Integration des Sinus ist vernachlässigbar (vgl. evaluation_t)

    return 0;
}
