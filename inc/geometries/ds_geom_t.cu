#include "hip/hip_runtime.h"

#include <iostream>
#include <iomanip>
#include <sstream>
#include <vector>
#include <string>
#include <fstream>

#include <cusp/print.h>

#include "file/nc_utilities.h"
#include "draw/host_window.h"
#include "dg/backend/timer.cuh"
#include "dg/backend/xspacelib.cuh"
#include "dg/algorithm.h"
#include "dg/poisson.h"
#include "dg/backend/functions.h"
#include "dg/backend/interpolation.cuh"

#include "solovev.h"
#include "init.h"
#include "testfunctors.h"
#include "magnetic_field.h"
#include "ds.h"

struct Parameters
{
    unsigned n, Nx, Ny, Nz;
    double boxscaleRm, boxscaleRp;
    double boxscaleZm, boxscaleZp;
    Parameters( const Json::Value& js){
        n = js["n"].asUInt();
        Nx = js["Nx"].asUInt();
        Ny = js["Ny"].asUInt();
        Nz = js.get("Nz", 1).asUInt();
        boxscaleRm = js.get("boxscaleRm", 1.).asDouble();
        boxscaleRp = js.get("boxscaleRp", 1.).asDouble();
        boxscaleZm = js.get("boxscaleZm", 1.3).asDouble();
        boxscaleZp = js.get("boxscaleZp", 1.).asDouble();
    }
    void display( std::ostream& os = std::cout ) const
    {
        os << "Input parameters are: \n";
        os  <<" n             = "<<n<<"\n"
            <<" Nx            = "<<Nx<<"\n"
            <<" Ny            = "<<Ny<<"\n"
            <<" Nz            = "<<Nz<<"\n"
            <<" boxscaleRm    = "<<boxscaleRm<<"\n"
            <<" boxscaleRp    = "<<boxscaleRp<<"\n"
            <<" boxscaleZm    = "<<boxscaleZm<<"\n"
            <<" boxscaleZp    = "<<boxscaleZp<<"\n";
        os << std::flush;
    }
};

struct InvNormR
{
    InvNormR( dg::geo::solovev::GeomParameters gp): R_0(gp.R_0){}
    double operator()( double R, double Z, double phi)const
    {
        return R_0/R;
    }
    
    private:
    double R_0;
}; 

//program seems to be flawed somehow at least I don't get it to work properly (MW) 
int main( int argc, char* argv[])
{
    if( !(argc == 3 ))
    {
        std::cerr << "ERROR: Wrong number of arguments!\n";
        std::cerr << " Usage: "<< argv[0]<<" [input.js] [geometry.js]\n";
        std::cerr << " ( Minimum input json file is { \"n\" : 3, \"Nx\": 100, \"Ny\":100 })\n";
        return -1;
    }
    std::string newfilename;
    Json::Reader reader;
    Json::Value input_js, geom_js;
    {
        std::cout << argv[0]<< " "<<argv[1]<<" & "<<argv[2]<<std::endl;
        std::ifstream isI( argv[1]);
        std::ifstream isG( argv[2]);
        reader.parse( isI, input_js, false);
        reader.parse( isG, geom_js, false);
    }
    const Parameters p(input_js);
    const dg::geo::solovev::GeomParameters gp(geom_js);
    p.display( std::cout);
    gp.display( std::cout);

    double Rmin=gp.R_0-p.boxscaleRm*gp.a;
    double Zmin=-p.boxscaleZm*gp.a*gp.elongation;
    double Rmax=gp.R_0+p.boxscaleRp*gp.a; 
    double Zmax=p.boxscaleZp*gp.a*gp.elongation;
    std::cout << "The grid parameters" <<"\n";
    std::cout  << Rmin<<" rho_s " << Rmax <<" rho_s " << Zmin <<" rho_s " <<Zmax <<" rho_s " <<"\n";

    dg::geo::TokamakMagneticField c = dg::geo::createSolovevField(gp);
        
    dg::geo::InvB invB(c);
    dg::geo::LnB lnB(c);
    dg::geo::BR bR(c);
    dg::geo::BZ bZ(c);
    dg::geo::CurvatureNablaBR curvatureR(c);
    dg::geo::CurvatureNablaBZ curvatureZ(c);
    dg::geo::GradLnB gradLnB(c);
    dg::geo::Pupil pupil(c.psip(), gp.psipmaxcut);
    InvNormR invnormr(gp);
    dg::geo::FieldR fieldR(c);
    dg::geo::FieldZ fieldZ(c);
    dg::geo::FieldP fieldP(c);
    dg::geo::BHatR bhatR(c);
    dg::geo::BHatZ bhatZ(c);
    dg::geo::BHatP bhatP(c);
    dg::DSFieldCylindrical field(dg::geo::BinaryVectorLvl0(bhatR, bhatZ, bhatP));
    dg::Grid3d grid( Rmin,Rmax, Zmin,Zmax, 0, 2.*M_PI,p.n, p.Nx, p.Ny,p.Nz);
    dg::HVec vecR = dg::evaluate( fieldR, grid);
    dg::HVec vecZ = dg::evaluate( fieldZ, grid);
    dg::HVec vecP = dg::evaluate( fieldP, grid);

    file::NC_Error_Handle err;
    int ncid, dim_ids[3];
//     err = nc_create( "geometry.nc", NC_CLOBBER, &ncid);
    err = nc_create( "geometry.nc", NC_NETCDF4|NC_CLOBBER, &ncid);

    err = file::define_dimensions( ncid, dim_ids, grid);
    int vecID[3];
    err = nc_def_var( ncid, "BR", NC_DOUBLE, 3, dim_ids, &vecID[0]);
    err = nc_def_var( ncid, "BZ", NC_DOUBLE, 3, dim_ids, &vecID[1]);
    err = nc_def_var( ncid, "BP", NC_DOUBLE, 3, dim_ids, &vecID[2]);
    err = nc_enddef( ncid);
    err = nc_put_var_double( ncid, vecID[0], vecR.data());
    err = nc_put_var_double( ncid, vecID[1], vecZ.data());
    err = nc_put_var_double( ncid, vecID[2], vecP.data());
    nc_close(ncid);
    std::cout << "-----(0) Check single field by integrating from 0 to 2pi (psi=0 surface)" << "\n";
    thrust::host_vector<double>  in(3);
    thrust::host_vector<double>  out(3);
    in[0]=gp.R_0+gp.a*0.6; 
    in[1]=0.0;
    in[2]=0.0;
    dg::integrateRK4( field, in, out,  2*M_PI, gp.rk4eps);
    
    std::cout <<"Rin =  "<< in[0] <<" Zin =  "<<in[1] <<" sin  = "<<in[2]<<"\n";
    std::cout <<"Rout = "<< out[0]<<" Zout = "<<out[1]<<" sout = "<<out[2]<<"\n";


    
    unsigned n=p.n, Nx=p.Nx, Ny=p.Ny, Nz=p.Nz;
    for (unsigned k=1;k<2;k++) //n iterator
    {
        for (unsigned i=0;i<1;i++) //Nxy iterator
        {
            std::stringstream ss1,ss2;
            ss1 << "dserr1n" <<k*n<<"Nxy"<<pow(2,i)* Nx<<".txt";
            ss2 << "dserr2n" <<k*n<<"Nxy"<<pow(2,i)* Nx<<".txt";
            std::string dserr1fn = ss1.str();
            std::string dserr2fn = ss2.str();
    //         std::cout << dserr1fn;
            std::ofstream dserrfile1((char *) dserr1fn.c_str());
            std::ofstream dserrfile2((char *) dserr2fn.c_str());
            for (unsigned zz=0;zz<1;zz++) //Nz iterator
            {
                std::cout << "n = " << k*n << " Nx = " <<pow(2,i)* Nx << " Ny = " <<pow(2,i)* Ny << " Nz = "<<pow(2,zz)* Nz <<"\n";
                //Similar to feltor grid
                dg::CylindricalGrid3d g3d( Rmin,Rmax, Zmin,Zmax, 0, 2.*M_PI,k*n,pow(2,i)* Nx,pow(2,i)* Ny, pow(2,zz)*Nz,dg::NEU, dg::NEU, dg::PER);
                const dg::DVec w3d = dg::create::volume( g3d);
                dg::DVec pupilongrid = dg::evaluate( pupil, g3d);

                std::cout <<"---------------------------------------------------------------------------------------------" << "\n";
                std::cout <<"-----(1a) test with testfunction  (works for DIR)" << "\n";
                dg::geo::TestFunction func(c);
                dg::geo::DeriTestFunction derifunc(c);
                std::cout << "Construct parallel  derivative\n";
                dg::Timer t;
                t.tic();
                dg::FieldAligned<dg::aGeometry3d, dg::IDMatrix, dg::DVec > dsFA( field, g3d, gp.rk4eps, dg::geo::PsiLimiter(c.psip(), gp.psipmaxlim), g3d.bcx()); 
                dg::DS<dg::aGeometry3d, dg::IDMatrix, dg::DMatrix, dg::DVec>  ds( dsFA, field, dg::normed, dg::centered); //choose bc of grid
                t.toc();
                std::cout << "-----> Creation of parallel Derivative took"<<t.diff()<<"s\n";

                dg::DVec function = dg::evaluate( func, g3d),dsfunc(function);
                dg::DVec diff(g3d.size());

                dg::DVec solution = dg::evaluate( derifunc, g3d);
                ds.set_boundaries( dg::PER, 0, 0);
                ds( function, dsfunc);
                //cut boundaries
//                 dg::blas1::pointwiseDot( pupilongrid, dsfunc, dsfunc);  //damped dsfunc
//                 dg::blas1::pointwiseDot( pupilongrid, solution, solution); //damped dssol

                dg::blas1::axpby( 1., solution, -1., dsfunc,diff);
                double normds = dg::blas2::dot( w3d, dsfunc);
                std::cout << "Norm ds  = "<<sqrt( normds)<<"\n";
                double normsol = dg::blas2::dot( w3d,solution);
                std::cout << "Norm sol = "<<sqrt( normsol)<<"\n";
                double normdiff = dg::blas2::dot( w3d, diff);
                double reldiff=sqrt( normdiff/normsol );
                std::cout << "Rel Diff = "<< reldiff<<"\n";
                  std::cout <<"-----(1b) test parallel derivative created brackets with testfunction  (works for DIR/NEU)" << "\n";
//                 solovev::TestFunction func(psip);
//                 solovev::DeriTestFunction derifunc(gp,psip,psipR,psipZ,ipol,invB);
                std::cout << "-----> Construct parallel  derivative\n";
                t.tic();
                dg::DVec dsRZPhifunction(g3d.size());
                dg::DVec dsR(g3d.size());
                dg::DVec dsZ(g3d.size());
                dg::DVec dsPHI(g3d.size());
                dg::DVec BvecR = dg::evaluate( bhatR, grid);
                dg::DVec BvecZ = dg::evaluate( bhatZ, grid);
                dg::DVec BvecPHI = dg::evaluate( bhatP, grid);

                dg::DMatrix dR   =dg::create::dx( g3d, g3d.bcx(),dg::centered);
                dg::DMatrix dZ   =dg::create::dy( g3d, g3d.bcy(),dg::centered);
                dg::DMatrix dPHI =dg::create::dz( g3d, g3d.bcz(),dg::centered);
                
                dg::blas2::symv( dR, function, dsR);  
                dg::blas2::symv( dZ, function,   dsZ); 
                dg::blas2::symv( dPHI,function, dsPHI); 
                dg::blas1::pointwiseDot(BvecR ,dsR,dsR); //BR*dR f
                dg::blas1::pointwiseDot(BvecZ ,dsZ,dsZ); //BZ*dZ f
                dg::blas1::pointwiseDot(BvecPHI ,dsPHI,dsPHI);//Bphi*dphi f
                
                dg::blas1::axpby(1.,dsR,1.,dsZ,dsRZPhifunction); //BR*dR f + BZ*dZ f
                dg::blas1::axpby(1.,dsPHI,1.,dsRZPhifunction,dsRZPhifunction); //BR*dR f + BZ*dZ f+Bphi*dphi f
                t.toc();

                std::cout << "-----> Creation of parallel Derivative took "<<t.diff()<<"s\n";

                dg::DVec diffRZPhi(g3d.size());

                //cut boundaries
//                 dg::blas1::pointwiseDot( pupilongrid,dsRZPhifunction, dsRZPhifunction);  //damped dsfunc
//                 dg::blas1::pointwiseDot( pupilongrid, solution, solution); //damped dssol

                dg::blas1::axpby( 1., solution, -1., dsRZPhifunction, diffRZPhi);
                double normdsRZPhi = dg::blas2::dot( w3d,  dsRZPhifunction);
                std::cout << "Norm dsRZPhi  = "<<sqrt( normdsRZPhi)<<"\n";
                std::cout << "Norm sol = "<<sqrt( normsol)<<"\n";
                double normdiffRZPhi = dg::blas2::dot( w3d, diffRZPhi);
                double reldiffRZPhi=sqrt( normdiffRZPhi/normsol );
                std::cout << "Rel Diff = "<< reldiffRZPhi<<"\n";
                
                std::cout <<"---------------------------------------------------------------------------------------------" << "\n";
                std::cout <<"-----(2a) test with gradlnb" << "\n";    
                dg::DVec gradLnBsolution = dg::evaluate( gradLnB, g3d);
                dg::DVec lnBongrid = dg::evaluate( lnB, g3d);
                dg::DVec dslnBongrid(g3d.size());
                dg::DVec diff2(g3d.size());
                dg::DVec pupilongradLnBsolution(gradLnBsolution);

                ds(lnBongrid,dslnBongrid);
                
                //cut boundaries
//                 dg::blas1::pointwiseDot( pupilongrid, dslnBongrid, dslnBongrid); 
//                 dg::blas1::pointwiseDot( pupilongrid, gradLnBsolution, pupilongradLnBsolution); 

                dg::blas1::axpby( 1., gradLnBsolution , -1., dslnBongrid,diff2); //diff = gradlnB - ds(ln(B))
                //cut boundaries

                double normds2 = dg::blas2::dot( w3d, dslnBongrid); //=  Integral (gds(ln(B))^2 )
                std::cout << "Norm ds  = "<<sqrt( normds2)<<"\n";
                double normsol2 = dg::blas2::dot( w3d,gradLnBsolution);//=  Integral (gradlnB^2 )
                std::cout << "Norm sol = "<<sqrt( normsol2)<<"\n";
                double normdiff2=dg::blas2::dot( w3d, diff2); //=  Integral ((gradlnB - ds(ln(B)))^2)
                double reldiff2 =sqrt( normdiff2/normsol2 ); ;//=  sqrt(Integral ((gradlnB - ds(ln(B)))^2)/Integral (gradlnB^2 ))
                std::cout << "Rel Diff = "<<reldiff2 <<"\n";
                std::cout <<"-----(2b) test with gradlnb" << "\n";    
                dg::DVec dsRlnB(g3d.size());
                dg::DVec dsZlnB(g3d.size());
                dg::DVec dsPHIlnB(g3d.size());
                dg::DVec dsRZPHIlnB(g3d.size());

                dg::DVec diff2b(g3d.size());
                dg::blas2::symv( dR, lnBongrid, dsRlnB);  
                dg::blas2::symv( dZ, lnBongrid,   dsZlnB); 
                dg::blas2::symv( dPHI,lnBongrid, dsPHIlnB); 
                dg::blas1::pointwiseDot(BvecR ,dsRlnB,dsRlnB); //BR*dR f
                dg::blas1::pointwiseDot(BvecZ ,dsZlnB,dsZlnB); //BZ*dZ f
                dg::blas1::pointwiseDot(BvecPHI ,dsPHIlnB,dsPHIlnB);//Bphi*dphi f
                
                dg::blas1::axpby(1.,dsRlnB,1.,dsZlnB,dsRZPHIlnB); //BR*dR f + BZ*dZ f
                dg::blas1::axpby(1.,dsPHIlnB,1.,dsRZPHIlnB,dsRZPHIlnB); //BR*dR f + BZ*dZ f+Bphi*dphi 
                
                //cut boundaries
//                 dg::blas1::pointwiseDot( pupilongrid, dslnBongrid, dslnBongrid); 
//                 dg::blas1::pointwiseDot( pupilongrid, gradLnBsolution, pupilongradLnBsolution); 

                dg::blas1::axpby( 1., gradLnBsolution , -1.,dsRZPHIlnB,diff2b); //diff = gradlnB - ds(ln(B))
                //cut boundaries

                double normds2b = dg::blas2::dot( w3d,dsRZPHIlnB); //=  Integral (gds(ln(B))^2 )
                std::cout << "Norm ds  = "<<sqrt( normds2b)<<"\n";
//                 double normsol2b = dg::blas2::dot( w3d,pupilongradLnBsolution);//=  Integral (gradlnB^2 )
                std::cout << "Norm sol = "<<sqrt( normsol2)<<"\n";
                double normdiff2b=dg::blas2::dot( w3d, diff2b); //=  Integral ((gradlnB - ds(ln(B)))^2)
                double reldiff2b =sqrt( normdiff2b/normsol2 ); ;//=  sqrt(Integral ((gradlnB - ds(ln(B)))^2)/Integral (gradlnB^2 ))
                std::cout << "Rel Diff = "<<reldiff2b <<"\n";
                std::cout <<"---------------------------------------------------------------------------------------------" << "\n";
                std::cout <<"-----(3) test with gradlnb and with (a) Arakawa and (b) Poisson discretization" << "\n";    
                dg::ArakawaX< dg::CylindricalGrid3d, dg::DMatrix, dg::DVec> arakawa(g3d); 
                dg::Poisson<  dg::CylindricalGrid3d, dg::DMatrix, dg::DVec> poiss(g3d);
                dg::DVec invBongrid = dg::evaluate( invB, g3d);
                dg::DVec psipongrid = dg::evaluate( c.psip(), g3d);
                dg::DVec invnormrongrid = dg::evaluate( invnormr, g3d);
                dg::DVec arakawasolution(g3d.size());
                dg::DVec poisssolution(g3d.size());
                dg::DVec diff3(g3d.size());
                dg::DVec diff4(g3d.size());
//                 dg::blas1::pointwiseDot( pupilongrid, invnormrongrid, invnormrongrid); 
//                 dg::blas1::pointwiseDot( pupilongrid, invBongrid, invBongrid); 

                arakawa( lnBongrid, psipongrid, arakawasolution); //1/B [B,psip]
                poiss(   lnBongrid, psipongrid, poisssolution); //1/B [B,psip]
                dg::blas1::pointwiseDot( invBongrid, arakawasolution, arakawasolution); //1/B^2 [B,psip]
                dg::blas1::pointwiseDot( invnormrongrid, arakawasolution, arakawasolution); //1/(R B^2) [B,psip]
//                 dg::blas1::pointwiseDot( pupilongrid, arakawasolution, arakawasolution); 

                dg::blas1::pointwiseDot( invBongrid, poisssolution, poisssolution); //    1/B^2 [B,psip]
                dg::blas1::pointwiseDot( invnormrongrid, poisssolution, poisssolution); //1/(R B^2) [B,psip]
//                 dg::blas1::pointwiseDot( pupilongrid, poisssolution, poisssolution); 

                
                dg::blas1::axpby( 1., pupilongradLnBsolution , -1., arakawasolution,diff3);

                double normarak= dg::blas2::dot( w3d, arakawasolution); //=  Integral (gds(ln(B))^2 )
                std::cout << "Norm normarak  = "<<sqrt( normarak)<<"\n";
                double normdiff3=dg::blas2::dot( w3d, diff3); //=  Integral ((gradlnB - ds(ln(B)))^2)
                double reldiff3 =sqrt( normdiff3/normsol2 ); ;//=  sqrt(Integral ((gradlnB - ds(ln(B)))^2)/Integral (gradlnB^2 ))
                std::cout << "Rel Diff = "<<reldiff3 <<"\n";
                
                dg::blas1::axpby( 1., pupilongradLnBsolution , -1., poisssolution,diff4);

                double normpoiss= dg::blas2::dot( w3d, poisssolution); //=  Integral (gds(ln(B))^2 )
                std::cout << "Norm normpoiss  = "<<sqrt( normpoiss)<<"\n";
                double normdiff4=dg::blas2::dot( w3d, diff4); //=  Integral ((gradlnB - ds(ln(B)))^2)
                double reldiff4 =sqrt( normdiff4/normsol2 ); ;//=  sqrt(Integral ((gradlnB - ds(ln(B)))^2)/Integral (gradlnB^2 ))
                std::cout << "Rel Diff = "<<reldiff4 <<"\n";
                
                dserrfile1 << pow(2,zz)*Nz <<" " << reldiff << std::endl;
                dserrfile2 << pow(2,zz)*Nz <<" " << reldiff2 << std::endl;
                std::cout <<"---------------------------------------------------------------------------------------------" << "\n";
                std::cout <<"----(4) test div(B) != 0 (works for NEU)"<<"\n";
                dg::DVec bRongrid = dg::evaluate( fieldR, grid);
                dg::DVec bZongrid = dg::evaluate( fieldZ, grid);
                dg::DVec dRbR(g3d.size());
                dg::DVec dZbZ(g3d.size());                
                dg::DVec invRbR(g3d.size());                
//              //cut boundaries
//                 dg::blas1::pointwiseDot( pupilongrid, bRongrid,bRongrid); 
//                 dg::blas1::pointwiseDot( pupilongrid, bZongrid, bZongrid); 
                invnormrongrid = dg::evaluate( invnormr, g3d);

                dg::DVec divB(g3d.size());                
//                 dg::blas2::gemv( arakawa.dx(), bRongrid, dRbR);
//                 dg::blas2::gemv( arakawa.dy(), bZongrid, dZbZ);
                dg::blas2::gemv( poiss.dxlhs(), bRongrid, dRbR); //d_R B^R
                dg::blas2::gemv( poiss.dylhs(), bZongrid, dZbZ); //d_Z B^Z
                dg::blas1::pointwiseDot( invnormrongrid , bRongrid, invRbR); // R_0/R B^R
                dg::blas1::axpby( 1., dRbR   , 1., dZbZ, divB); //d_R B^R + d_Z B^Z
                dg::blas1::axpby( 1./gp.R_0, invRbR , 1., divB); //( B^R/R/R_0 + d_R B^R + d_Z B^Z)
                dg::blas1::pointwiseDot( pupilongrid, divB, divB);  //cut 

                double normdivB2= dg::blas2::dot( w3d, divB); 
                std::cout << "divB = "<<sqrt( normdivB2)<<"\n";
                std::cout <<"---------------------------------------------------------------------------------------------" << "\n";
                std::cout <<"----(5) test grad_par (psi_p) != 0 (works for NEU)"<<"\n";
                dg::DVec dspsi(g3d.size());
                ds( psipongrid, dspsi);
                double normdspsi = dg::blas2::dot( w3d, dspsi);
                std::cout << "Norm grad_par (psi_p)  = "<<sqrt( normdspsi)<<"\n";
       
             }
            dserrfile1.close();
            dserrfile2.close();
   
        }

    }
    

    return 0;
}
