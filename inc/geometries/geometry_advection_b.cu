#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>

#include "dg/arakawa.h"
#include "dg/poisson.h"
#include "dg/geometry/geometry.h"

#include "curvilinear.h"

#include "flux.h"
#include "simple_orthogonal.h"

#include "solovev.h"
#include "magnetic_field.h"
#include "testfunctors.h"
#include "dg/backend/timer.cuh"

struct FuncDirPer2
{
    FuncDirPer2( dg::geo::TokamakMagneticField c, double psi_0, double psi_1):
        R_0_(c.R0()), psi0_(psi_0), psi1_(psi_1), psip_(c.psip()), psipR_(c.psipR()), psipZ_(c.psipZ()){}
    double operator()(double R, double Z, double phi) const {
        return this->operator()(R,Z);
    }
    double operator()(double R, double Z) const {
        double psip = psip_.get()(R,Z);
        return (psip-psi0_)*(psip-psi1_)*cos(theta(R,Z));
    }
    double dR( double R, double Z)const
    {
        double psip = psip_.get()(R,Z), psipR = psipR_.get()(R,Z), theta_ = theta(R,Z);
        return (2.*psip*psipR - (psi0_+psi1_)*psipR)*cos(theta_) 
            - (psip-psi0_)*(psip-psi1_)*sin(theta_)*thetaR(R,Z);
    }
    double dZ( double R, double Z)const
    {
        double psip = psip_.get()(R,Z), psipZ = psipZ_.get()(R,Z), theta_=theta(R,Z);
        return (2*psip*psipZ - (psi0_+psi1_)*psipZ)*cos(theta_) 
            - (psip-psi0_)*(psip-psi1_)*sin(theta_)*thetaZ(R,Z);
    }
    private:
    double theta( double R, double Z) const {
        double dR = R-R_0_;
        if( Z >= 0)
            return acos( dR/sqrt( dR*dR + Z*Z));
        else
            return 2.*M_PI-acos( dR/sqrt( dR*dR + Z*Z));
    }
    double thetaR( double R, double Z) const {
        double dR = R-R_0_;
        return -Z/(dR*dR+Z*Z);
    }
    double thetaZ( double R, double Z) const {
        double dR = R-R_0_;
        return dR/(dR*dR+Z*Z);
    }
    double R_0_;
    double psi0_, psi1_;
    dg::Handle<dg::geo::aBinaryFunctor> psip_, psipR_,  psipZ_;
};

struct ArakawaDirPer
{
    ArakawaDirPer( dg::geo::TokamakMagneticField c, double psi_0, double psi_1): 
        f_(c, psi_0, psi_1, 4), g_(c, psi_0, psi_1){ }
    double operator()(double R, double Z, double phi) const {
        return this->operator()(R,Z);
    }
    double operator()(double R, double Z) const {
        return f_.dR( R,Z)*g_.dZ(R,Z) - f_.dZ(R,Z)*g_.dR(R,Z);
    }
    private:
    dg::geo::FuncDirPer f_;
    FuncDirPer2 g_;
};

struct VariationDirPer
{
    VariationDirPer( dg::geo::TokamakMagneticField c, double psi_0, double psi_1): f_(c, psi_0, psi_1,4. ){}
    double operator()(double R, double Z, double phi) const {
        return this->operator()(R,Z);}

    double operator()(double R, double Z) const {
        return f_.dR( R,Z)*f_.dR(R,Z) + f_.dZ(R,Z)*f_.dZ(R,Z);
    }
    private:
    dg::geo::FuncDirPer f_;
};

struct CurvatureDirPer
{
    CurvatureDirPer( dg::geo::TokamakMagneticField c, double psi_0, double psi_1): f_(c, psi_0, psi_1,4.), curvR(c), curvZ(c){}
    double operator()(double R, double Z, double phi) const {
        return this->operator()(R,Z);}
    double operator()(double R, double Z) const {
        return curvR( R,Z)*f_.dR(R,Z) + curvZ(R,Z)*f_.dZ(R,Z);
    }
    private:
    dg::geo::FuncDirPer f_;
    dg::geo::CurvatureNablaBR curvR;
    dg::geo::CurvatureNablaBZ curvZ;
};



int main(int argc, char** argv)
{
    std::cout << "Type n, Nx, Ny\n";
    unsigned n, Nx, Ny;
    std::cin >> n>> Nx>>Ny;   
    Json::Reader reader;
    Json::Value js;
    if( argc==1)
    {
        std::ifstream is("geometry_params_Xpoint.js");
        reader.parse(is,js,false);
    }
    else
    {
        std::ifstream is(argv[1]);
        reader.parse(is,js,false);
    }
    dg::geo::solovev::GeomParameters gp(js);
    dg::geo::TokamakMagneticField c = dg::geo::createSolovevField( gp);

    std::cout << "Psi min "<<c.psip()(gp.R_0, 0)<<"\n";
    std::cout << "Type psi_0 (-20) and psi_1 (-4)\n";
    double psi_0, psi_1;
    std::cin >> psi_0>> psi_1;
    std::cout << "Psi_0 = "<<psi_0<<" psi_1 = "<<psi_1<<std::endl;
    //gp.display( std::cout);
    dg::Timer t;
    //solovev::detail::Fpsi fpsi( gp, -10);
    std::cout << "Constructing grid ... \n";
    t.tic();
    //dg::geo::RibeiroFluxGenerator ribeiro( c.get_psip(), psi_0, psi_1, gp.R_0, 0., 1);
    dg::geo::FluxGenerator ribeiro( c.get_psip(), c.get_ipol(), psi_0, psi_1, gp.R_0, 0., 1);
    //dg::geo::SimpleOrthogonal ribeiro( c.get_psip(), psi_0, psi_1, gp.R_0, 0., 1);
    dg::CurvilinearGrid2d grid(ribeiro, n, Nx, Ny, dg::DIR); //2d
    t.toc();
    std::cout << "Construction took "<<t.diff()<<"s"<<std::endl;
    grid.display();

    const dg::DVec vol = dg::create::volume( grid);
    std::cout <<std::fixed<< std::setprecision(6)<<std::endl;


    dg::geo::FuncDirPer left(c, psi_0, psi_1, 4);
    FuncDirPer2 right( c, psi_0, psi_1);
    ArakawaDirPer jacobian( c, psi_0, psi_1);
    VariationDirPer variationLHS(c, psi_0, psi_1);

    const dg::DVec lhs = dg::pullback( left, grid);
    dg::DVec jac(lhs);
    const dg::DVec rhs = dg::pullback( right, grid);
    const dg::DVec sol = dg::pullback ( jacobian, grid);
    const dg::DVec variation = dg::pullback ( variationLHS, grid);
    dg::DVec eins = dg::evaluate( dg::one, grid);

    ///////////////////////////////////////////////////////////////////////
    std::cout << "TESTING ARAKAWA\n";
    dg::ArakawaX<dg::aGeometry2d, dg::DMatrix, dg::DVec> arakawa( grid);
    arakawa( lhs, rhs, jac);
    const double norm = dg::blas2::dot( sol, vol, sol);
    std::cout << std::scientific;
    double result = dg::blas2::dot( eins, vol, jac);
    std::cout << "Mean     Jacobian is "<<result<<"\n";
    result = dg::blas2::dot( rhs, vol, jac);
    std::cout << "Mean rhs*Jacobian is "<<result<<"\n";
    result = dg::blas2::dot( lhs, vol, jac);
    std::cout << "Mean lhs*Jacobian is "<<result<<"\n";
    //std::cout << "norm of solution "<<norm<<"\n";
    //std::cout << "norm of Jacobian "<<dg::blas2::dot( jac, vol, jac)<<"\n";
    //std::cout << "norm of lhs      "<<dg::blas2::dot( lhs, vol, lhs)<<"\n";
    //std::cout << "norm of rhs      "<<dg::blas2::dot( rhs, vol, rhs)<<"\n";
    dg::blas1::axpby( 1., sol, -1., jac);
    result = dg::blas2::dot( jac, vol, jac);
    std::cout << "          Rel. distance to solution "<<sqrt( result/norm)<<std::endl; //don't forget sqrt when comuting errors
    arakawa.variation( lhs, jac);
    const double normVar = dg::blas2::dot( vol, variation);
    //std::cout << "norm of variation "<<normVar<<"\n";
    dg::blas1::axpby( 1., variation, -1., jac);
    result = dg::blas2::dot( jac, vol, jac);
    std::cout << "Variation rel. distance to solution "<<sqrt( result/normVar)<<std::endl; //don't forget sqrt when comuting errors
    ///////////////////////////////////////////////////////////////////////
    std::cout << "TESTING POISSON\n";
    dg::Poisson<dg::aGeometry2d, dg::DMatrix, dg::DVec> poisson( grid);
    poisson( lhs, rhs, jac);
    result = dg::blas2::dot( eins, vol, jac);
    std::cout << "Mean     Jacobian is "<<result<<"\n";
    result = dg::blas2::dot( rhs, vol, jac);
    std::cout << "Mean rhs*Jacobian is "<<result<<"\n";
    result = dg::blas2::dot( lhs, vol, jac);
    std::cout << "Mean lhs*Jacobian is "<<result<<"\n";
    result = dg::blas2::dot( jac, vol, jac);
    //std::cout << "norm of solution "<<norm<<"\n";
    //std::cout << "norm of Jacobian "<<result<<"\n";
    dg::blas1::axpby( 1., sol, -1., jac);
    result = dg::blas2::dot( jac, vol, jac);
    std::cout << "          Rel. distance to solution "<<sqrt( result/norm)<<std::endl; //don't forget sqrt when comuting errors
    poisson.variationRHS( lhs, jac);
    dg::blas1::axpby( 1., variation, -1., jac);
    result = dg::blas2::dot( jac, vol, jac);
    std::cout << "Variation rel. distance to solution "<<sqrt( result/normVar)<<std::endl; //don't forget sqrt when comuting errors

    ////////////////////////////transform curvature components////////
    std::cout << "TESTING CURVATURE 3D\n";
    dg::DVec curvX, curvY;
    dg::HVec tempX, tempY;
    dg::pushForwardPerp(dg::geo::CurvatureNablaBR(c), dg::geo::CurvatureNablaBZ(c), tempX, tempY, grid);
    dg::blas1::transfer(  tempX, curvX);
    dg::blas1::transfer(  tempY, curvY);
    dg::DMatrix dx, dy;
    dg::blas2::transfer( dg::create::dx(grid), dx);
    dg::blas2::transfer( dg::create::dy(grid), dy);
    dg::DVec tempx(curvX), tempy(curvX);
    dg::blas2::symv( dx, lhs, tempx);
    dg::blas2::symv( dy, lhs, tempy);
    dg::blas1::pointwiseDot( tempx, curvX, tempx);
    dg::blas1::pointwiseDot( 1., tempy, curvY, 1.,  tempx);
    const double normCurv = dg::blas2::dot( tempx, vol, tempx);

    CurvatureDirPer curv(c, psi_0, psi_1);
    dg::DVec curvature;
    dg::blas1::transfer( dg::pullback(curv, grid), curvature);

    dg::blas1::axpby( 1., tempx, -1., curvature, tempx);
    result = dg::blas2::dot( vol, tempx);
    std::cout << "Curvature rel. distance to solution "<<sqrt( result/normCurv)<<std::endl; //don't forget sqrt when comuting errors



    return 0;
}
