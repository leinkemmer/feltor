#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <fstream>
#include <sstream>
#include <cmath>

#include "json/json.h"

#include "dg/backend/xspacelib.cuh"
#include "dg/functors.h"
#include "file/nc_utilities.h"

#include "solovev.h"
#include "taylor.h"
#include "init.h"
#include "magnetic_field.h"
#include "average.h"

struct Parameters
{
    unsigned n, Nx, Ny, Nz;
    double boxscaleRm, boxscaleRp;
    double boxscaleZm, boxscaleZp;
    double amp, k_psi, bgprofamp, nprofileamp;
    double sigma, posX, posY;
    Parameters( const Json::Value& js){
        n = js["n"].asUInt();
        Nx = js["Nx"].asUInt();
        Ny = js["Ny"].asUInt();
        Nz = js.get("Nz", 1).asUInt();
        boxscaleRm = js.get("boxscaleRm", 1.).asDouble();
        boxscaleRp = js.get("boxscaleRp", 1.).asDouble();
        boxscaleZm = js.get("boxscaleZm", 1.3).asDouble();
        boxscaleZp = js.get("boxscaleZp", 1.).asDouble();
        amp = js.get("amplitude", 1.).asDouble();
        k_psi = js.get("k_psi", 1.).asDouble();
        bgprofamp = js.get("bgprofamp", 1.).asDouble();
        nprofileamp = js.get("nprofileamp", 1.).asDouble();
        sigma = js.get("sigma", 10).asDouble();
        posX = js.get("posX", 0.5).asDouble();
        posY = js.get("posY", 0.5).asDouble();
    }
    void display( std::ostream& os = std::cout ) const
    {
        os << "Input parameters are: \n";
        os  <<" n             = "<<n<<"\n"
            <<" Nx            = "<<Nx<<"\n"
            <<" Ny            = "<<Ny<<"\n"
            <<" Nz            = "<<Nz<<"\n"
            <<" boxscaleRm    = "<<boxscaleRm<<"\n"
            <<" boxscaleRp    = "<<boxscaleRp<<"\n"
            <<" boxscaleZm    = "<<boxscaleZm<<"\n"
            <<" boxscaleZp    = "<<boxscaleZp<<"\n"
            <<" amp           = "<<amp<<"\n"
            <<" k_psi         = "<<k_psi<<"\n"
            <<" bgprofamp     = "<<bgprofamp<<"\n"
            <<" nprofileamp   = "<<nprofileamp<<"\n"
            <<" sigma         = "<<sigma<<"\n"
            <<" posX          = "<<posX<<"\n"
            <<" posY          = "<<posY<<"\n";
        os << std::flush;
    }
};

int main( int argc, char* argv[])
{
    if( !(argc == 4 || argc == 3))
    {
        std::cerr << "ERROR: Wrong number of arguments!\n";
        std::cerr << " Usage: "<< argv[0]<<" [input.js] [geom.js] [output.nc]\n";
        std::cerr << " ( Minimum input json file is { \"n\" : 3, \"Nx\": 100, \"Ny\":100 })\n";
        std::cerr << "Or \n Usage: "<< argv[0]<<" [file.nc] [output.nc]\n";
        std::cerr << " ( Program searches for string variables 'inputfile' and 'geomfile' in file.nc and tries a json parser)\n";
        return -1;
    }
    std::string newfilename;
    Json::Reader reader;
    Json::Value input_js, geom_js;
    if( argc == 4) 
    {
        newfilename = argv[3];
        std::cout << argv[0]<< " "<<argv[1]<<" & "<<argv[2]<<" -> " <<argv[3]<<std::endl;
        std::ifstream isI( argv[1]);
        std::ifstream isG( argv[2]);
        reader.parse( isI, input_js, false);
        reader.parse( isG, geom_js, false);
    }
    else
    {
        newfilename = argv[2];
        std::cout << argv[0]<< " "<<argv[1]<<" -> " <<argv[2]<<std::endl;
        //////////////////////////open nc file//////////////////////////////////
        file::NC_Error_Handle err;
        int ncid;
        err = nc_open( argv[1], NC_NOWRITE, &ncid);
        ///////////////read in and show inputfile und geomfile//////////////////
        std::string input, geom;
        size_t length;
        err = nc_inq_attlen( ncid, NC_GLOBAL, "inputfile", &length);
        input.resize( length, 'x');
        err = nc_get_att_text( ncid, NC_GLOBAL, "inputfile", &input[0]);
        err = nc_inq_attlen( ncid, NC_GLOBAL, "geomfile", &length);
        geom.resize( length, 'x');
        err = nc_get_att_text( ncid, NC_GLOBAL, "geomfile", &geom[0]);
        nc_close( ncid);
        reader.parse( input, input_js, false);
        reader.parse( geom, geom_js, false);
    }
    const Parameters p(input_js);
    const dg::geo::solovev::GeomParameters gp(geom_js);
    p.display( std::cout);
    gp.display( std::cout);
    std::string input = input_js.toStyledString();
    std::string geom = geom_js.toStyledString();
    unsigned n, Nx, Ny, Nz;
    n = p.n, Nx = p.Nx, Ny = p.Ny, Nz = p.Nz;
    double Rmin=gp.R_0-p.boxscaleRm*gp.a;
    double Zmin=-p.boxscaleZm*gp.a*gp.elongation;
    double Rmax=gp.R_0+p.boxscaleRp*gp.a; 
    double Zmax=p.boxscaleZp*gp.a*gp.elongation;
 
    //construct all geometry quantities
    dg::geo::TokamakMagneticField c = dg::geo::createTaylorField(gp);
    const double R_X = gp.R_0-1.1*gp.triangularity*gp.a;
    const double Z_X = -1.1*gp.elongation*gp.a;
    const double R_H = gp.R_0-gp.triangularity*gp.a;
    const double Z_H = gp.elongation*gp.a;
    const double alpha_ = asin(gp.triangularity);
    const double N1 = -(1.+alpha_)/(gp.a*gp.elongation*gp.elongation)*(1.+alpha_);
    const double N2 =  (1.-alpha_)/(gp.a*gp.elongation*gp.elongation)*(1.-alpha_);
    const double N3 = -gp.elongation/(gp.a*cos(alpha_)*cos(alpha_));
    std::cout << "TEST ACCURACY OF PSI\n";
    std::cout << "psip( 1+e,0)           "<<c.psip()(gp.R_0 + gp.a, 0.)<<"\n";
    std::cout << "psip( 1-e,0)           "<<c.psip()(gp.R_0 - gp.a, 0.)<<"\n";
    std::cout << "psip( 1-de,ke)         "<<c.psip()(R_H, Z_H)<<"\n";
    std::cout << "psip( 1-1.1de,-1.1ke)  "<<c.psip()(R_X, Z_X)<<"\n";
    std::cout << "psipZ( 1+e,0)          "<<c.psipZ()(gp.R_0 + gp.a, 0.)<<"\n";
    std::cout << "psipZ( 1-e,0)          "<<c.psipZ()(gp.R_0 - gp.a, 0.)<<"\n";
    std::cout << "psipR( 1-de,ke)        "<<c.psipR()(R_H,Z_H)<<"\n";
    std::cout << "psipR( 1-1.1de,-1.1ke) "<<c.psipR()(R_X,Z_X)<<"\n";
    std::cout << "psipZ( 1-1.1de,-1.1ke) "<<c.psipZ()(R_X,Z_X)<<"\n";
    std::cout << "psipZZ( 1+e,0)         "<<c.psipZZ()(gp.R_0+gp.a,0.)+N1*c.psipR()(gp.R_0+gp.a,0)<<"\n";
    std::cout << "psipZZ( 1-e,0)         "<<c.psipZZ()(gp.R_0-gp.a,0.)+N2*c.psipR()(gp.R_0-gp.a,0)<<"\n";
    std::cout << "psipRR( 1-de,ke)       "<<c.psipRR()(R_H,Z_H)+N3*c.psipZ()(R_H,Z_H)<<"\n";

    //Feltor quantities
    dg::geo::InvB invB(c);
    dg::geo::BR bR(c);
    dg::geo::BZ bZ(c);
    dg::geo::CurvatureNablaBR curvatureR(c);
    dg::geo::CurvatureNablaBZ curvatureZ(c);
    dg::geo::GradLnB gradLnB(c);
    dg::geo::FieldR  field(c);
    dg::geo::FieldR fieldR(c);
    dg::geo::FieldZ fieldZ(c);
    dg::geo::FieldP fieldP(c);
    dg::geo::Iris iris( c.psip(), gp.psipmin, gp.psipmax );
    dg::geo::Pupil pupil(c.psip(), gp.psipmaxcut);
    dg::geo::GaussianDamping dampgauss(c.psip(), gp.psipmaxcut, gp.alpha);
    dg::geo::GaussianProfDamping dampprof(c.psip(),gp.psipmax, gp.alpha);
    dg::geo::ZonalFlow zonalflow(p.amp, p.k_psi, gp, c.psip());
    dg::geo::PsiLimiter psilimiter(c.psip(), gp.psipmaxlim);
    dg::geo::Nprofile prof(p.bgprofamp, p.nprofileamp, gp, c.psip());

    dg::BathRZ bath(16,16,p.Nz,Rmin,Zmin, 30.,5.,p.amp);
//     dg::Gaussian3d bath(gp.R_0+p.posX*gp.a, p.posY*gp.a, M_PI, p.sigma, p.sigma, p.sigma, p.amp);
    dg::Gaussian3d blob(gp.R_0+p.posX*gp.a, p.posY*gp.a, M_PI, p.sigma, p.sigma, p.sigma, p.amp);
    dg::Grid2d grid2d(Rmin,Rmax,Zmin,Zmax, n,Nx,Ny);

    std::vector<dg::HVec> hvisual(21);
        //allocate mem for visual
    std::vector<dg::HVec> visual(21);

    //B field functions
    hvisual[1] = dg::evaluate( c.psip(), grid2d);
    hvisual[2] = dg::evaluate( c.ipol(), grid2d);
    hvisual[3] = dg::evaluate( invB, grid2d);
    hvisual[4] = dg::evaluate( field, grid2d);
    hvisual[5] = dg::evaluate( curvatureR, grid2d);
    hvisual[6] = dg::evaluate( curvatureZ, grid2d);
    hvisual[7] = dg::evaluate( gradLnB, grid2d);
    //cut functions
    hvisual[8] = dg::evaluate( iris, grid2d);
    hvisual[9] = dg::evaluate( pupil, grid2d);
    hvisual[10] = dg::evaluate( dampprof, grid2d);
    hvisual[11] = dg::evaluate( dampgauss, grid2d);
    hvisual[12] = dg::evaluate( psilimiter, grid2d);
    //initial functions
    hvisual[13] = dg::evaluate( zonalflow, grid2d);
    hvisual[14] = dg::evaluate( prof, grid2d);
    hvisual[15] = dg::evaluate( blob, grid2d);
    hvisual[16] = dg::evaluate( bath, grid2d);
    //initial functions damped and with profile
    hvisual[17] = dg::evaluate( dg::one, grid2d);
    hvisual[18] = dg::evaluate( dg::one, grid2d);
    hvisual[19] = dg::evaluate( dg::one, grid2d);
    hvisual[20] = dg::evaluate( dg::one, grid2d);            
    dg::blas1::axpby( 1.,hvisual[16] , 1.,hvisual[14],hvisual[17]); //prof + bath
    dg::blas1::axpby( 1.,hvisual[13] , 1.,hvisual[14],hvisual[18]); //prof + zonal
    dg::blas1::axpby( 1.,hvisual[15] , 1.,hvisual[14],hvisual[19]); //prof + blob
    dg::blas1::plus(hvisual[17], -1); //to n -1
    dg::blas1::plus(hvisual[18], -1); //to n -1
    dg::blas1::plus(hvisual[19], -1); //to n -1
    dg::blas1::pointwiseDot(hvisual[10], hvisual[17], hvisual[17]); //damped 
    dg::blas1::pointwiseDot(hvisual[10], hvisual[18], hvisual[18]); //damped 
    dg::blas1::pointwiseDot(hvisual[10], hvisual[19], hvisual[19]); //damped 

    //Compute flux average
    dg::geo::Alpha alpha(c); // = B^phi / |nabla psip |
    dg::DVec psipog2d   = dg::evaluate( c.psip(), grid2d);
    dg::DVec alphaog2d  = dg::evaluate( alpha, grid2d); 
    double psipmin = (float)thrust::reduce( psipog2d .begin(), psipog2d .end(), 0.0,thrust::minimum<double>()  );
    unsigned npsi = 3, Npsi = 150;//set number of psivalues
    psipmin += (gp.psipmax - psipmin)/(double)Npsi; //the inner value is not good
    dg::Grid1d grid1d(psipmin , gp.psipmax, npsi ,Npsi,dg::DIR);
    dg::geo::SafetyFactor< dg::DVec>     qprof(grid2d, c, alphaog2d );
    dg::HVec sf         = dg::evaluate( qprof,    grid1d);
    dg::HVec abs        = dg::evaluate( dg::cooX1d, grid1d);

    
    std::string names[] = { "", "psip", "ipol", "invB","invbf", "KR", 
                            "KZ", "gradLnB", "iris", "pupil", "dampprof", 
                            "damp", "lim",  "zonal", "prof", "blob", 
                            "bath", "ini1","ini2","ini3","ini4"};

    /////////////////////////////set up netcdf/////////////////////////////////////
    file::NC_Error_Handle err;
    int ncid;
    err = nc_create( newfilename.data(), NC_NETCDF4|NC_CLOBBER, &ncid);
    err = nc_put_att_text( ncid, NC_GLOBAL, "inputfile", input.size(), input.data());
    err = nc_put_att_text( ncid, NC_GLOBAL, "geomfile", geom.size(), geom.data());
    int dim1d_ids[1], dim2d_ids[2], dim3d_ids[3] ;
    err = file::define_dimension( ncid,"psi", &dim1d_ids[0], grid1d);
    dg::Grid3d grid3d(Rmin,Rmax,Zmin,Zmax, 0, 2.*M_PI, n,Nx,Ny,Nz);
    err = file::define_dimensions( ncid, &dim3d_ids[0], grid3d);
    dim2d_ids[0] = dim3d_ids[1], dim2d_ids[1] = dim3d_ids[2]; 

    //write 1d vectors
    int avgID[2];
    err = nc_def_var( ncid, "q-profile", NC_DOUBLE, 1, &dim1d_ids[0], &avgID[0]);
    err = nc_def_var( ncid, "psip1d", NC_DOUBLE, 1, &dim1d_ids[0], &avgID[1]);
    err = nc_enddef( ncid);
    err = nc_put_var_double( ncid, avgID[0], sf.data());
    err = nc_put_var_double( ncid, avgID[1], abs.data());
    err = nc_redef(ncid);

    //write 2d vectors
    for(unsigned i=1; i<hvisual.size(); i++)
    {
        int vectorID[1];
        err = nc_def_var( ncid, names[i].data(), NC_DOUBLE, 2, &dim2d_ids[0], &vectorID[0]);
        err = nc_enddef( ncid);
        err = nc_put_var_double( ncid, vectorID[0], hvisual[i].data());
        err = nc_redef(ncid);

    }
    //compute & write 3d vectors
    dg::HVec vecR = dg::evaluate( fieldR, grid3d);
    dg::HVec vecZ = dg::evaluate( fieldZ, grid3d);
    dg::HVec vecP = dg::evaluate( fieldP, grid3d);
    int vecID[3];
    err = nc_def_var( ncid, "BR", NC_DOUBLE, 3, &dim3d_ids[0], &vecID[0]);
    err = nc_def_var( ncid, "BZ", NC_DOUBLE, 3, &dim3d_ids[0], &vecID[1]);
    err = nc_def_var( ncid, "BP", NC_DOUBLE, 3, &dim3d_ids[0], &vecID[2]);
    err = nc_enddef( ncid);
    err = nc_put_var_double( ncid, vecID[0], vecR.data());
    err = nc_put_var_double( ncid, vecID[1], vecZ.data());
    err = nc_put_var_double( ncid, vecID[2], vecP.data());
    err = nc_redef(ncid);
    //////////////////////////////Finalize////////////////////////////////////
    err = nc_close(ncid);


    return 0;
}
