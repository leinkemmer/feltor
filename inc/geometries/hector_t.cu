#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <fstream>
#include <sstream>
#include <cmath>

#include "dg/backend/xspacelib.cuh"
#include "dg/functors.h"
#include "curvilinear.h"

#include "dg/backend/timer.cuh"
//#include "guenther.h"
#include "solovev.h"
#include "hector.h"
//#include "refined_conformal.h"
#include "init.h"

#include "file/nc_utilities.h"

using namespace dg::geo;

thrust::host_vector<double> periodify( const thrust::host_vector<double>& in, const dg::Grid2d& g)
{
    thrust::host_vector<double> out(g.size());
    for( unsigned i=0; i<g.Ny()-1; i++)
    for( unsigned k=0; k<g.n(); k++)
    for( unsigned j=0; j<g.Nx(); j++)
    for( unsigned l=0; l<g.n(); l++)
        out[((i*g.n() + k)*g.Nx() + j)*g.n()+l] = 
            in[((i*g.n() + k)*g.Nx() + j)*g.n()+l];
    for( unsigned i=g.Ny()-1; i<g.Ny(); i++)
    for( unsigned k=0; k<g.n(); k++)
    for( unsigned j=0; j<g.Nx(); j++)
    for( unsigned l=0; l<g.n(); l++)
        out[((i*g.n() + k)*g.Nx() + j)*g.n()+l] = 
            in[((0*g.n() + k)*g.Nx() + j)*g.n()+l];
    return out;
}

int main( int argc, char* argv[])
{
    std::cout << "Type nHector, NxHector, NyHector ( 13 2 10)\n";
    unsigned nGrid, NxGrid, NyGrid;
    std::cin >> nGrid>> NxGrid>>NyGrid;   
    std::cout << "Type epsHector (1e-10)\n";
    double epsHector;
    std::cin >> epsHector;
    std::cout << "Type n, Nx, Ny, Nz ( 3 4 40 1)\n";
    unsigned n, Nx, Ny, Nz;
    std::cin >> n>> Nx>>Ny>>Nz;   
    Json::Reader reader;
    Json::Value js;
    if( argc==1)
    {
        std::ifstream is("geometry_params_Xpoint.js");
        reader.parse(is,js,false);
    }
    else
    {
        std::ifstream is(argv[1]);
        reader.parse(is,js,false);
    }
    //write parameters from file into variables
    dg::geo::solovev::GeomParameters gp(js);
    dg::geo::BinaryFunctorsLvl2 psip = dg::geo::solovev::createPsip( gp); 
    std::cout << "Psi min "<<psip.f()(gp.R_0, 0)<<"\n";
    std::cout << "Type psi_0 and psi_1\n";
    double psi_0, psi_1;
    std::cin >> psi_0>> psi_1;
    gp.display( std::cout);
    dg::Timer t;
    //solovev::detail::Fpsi fpsi( gp, -10);
    std::cout << "Constructing conformal grid ... \n";
    t.tic();
    Hector<dg::IDMatrix, dg::DMatrix, dg::DVec>* hector;
    //%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
    int construction = 0;
    if( construction == 0)
    {
        hector = new Hector<dg::IDMatrix, dg::DMatrix, dg::DVec>( psip, psi_0, psi_1, gp.R_0, 0., nGrid, NxGrid, NyGrid, epsHector, true);
    }
    else if( construction == 1)
    {
        dg::geo::BinaryFunctorsLvl1 nc = dg::geo::make_NablaPsiInvCollective( psip);
        hector = new Hector<dg::IDMatrix, dg::DMatrix, dg::DVec>( psip, nc, psi_0, psi_1, gp.R_0, 0., nGrid, NxGrid, NyGrid, epsHector, true);
    }
    else
    {
        dg::geo::BinarySymmTensorLvl1 lc = dg::geo::make_LiseikinCollective( psip, 0.1, 0.001);
        hector = new Hector<dg::IDMatrix, dg::DMatrix, dg::DVec>( psip,lc, psi_0, psi_1, gp.R_0, 0., nGrid, NxGrid, NyGrid, epsHector, true);
    }
    //%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
    dg::CurvilinearProductGrid3d g3d(*hector, n, Nx, Ny,Nz, dg::DIR);
    dg::CurvilinearGrid2d g2d = g3d.perp_grid();

    dg::Grid2d g2d_periodic(g2d.x0(), g2d.x1(), g2d.y0(), g2d.y1(), g2d.n(), g2d.Nx(), g2d.Ny()+1); 
    t.toc();
    std::cout << "Construction took "<<t.diff()<<"s"<<std::endl;
    std::cout << "Length in u is    "<<hector->width()<<std::endl;
    int ncid;
    file::NC_Error_Handle err;
    err = nc_create( "conformal.nc", NC_NETCDF4|NC_CLOBBER, &ncid);
    int dim3d[2];
    err = file::define_dimensions(  ncid, dim3d, g2d_periodic);
    int coordsID[2], onesID, defID, confID,volID,divBID;
    err = nc_def_var( ncid, "x_XYP", NC_DOUBLE, 2, dim3d, &coordsID[0]);
    err = nc_def_var( ncid, "y_XYP", NC_DOUBLE, 2, dim3d, &coordsID[1]);
    //err = nc_def_var( ncid, "z_XYP", NC_DOUBLE, 3, dim3d, &coordsID[2]);
    err = nc_def_var( ncid, "psi", NC_DOUBLE, 2, dim3d, &onesID);
    err = nc_def_var( ncid, "deformation", NC_DOUBLE, 2, dim3d, &defID);
    err = nc_def_var( ncid, "error", NC_DOUBLE, 2, dim3d, &confID);
    err = nc_def_var( ncid, "volume", NC_DOUBLE, 2, dim3d, &volID);
    err = nc_def_var( ncid, "divB", NC_DOUBLE, 2, dim3d, &divBID);

    thrust::host_vector<double> psi_p = dg::pullback( psip.f(), g2d);
    //g.display();
    err = nc_put_var_double( ncid, onesID, periodify(psi_p, g2d_periodic).data());
    dg::HVec X( g2d.size()), Y(X); //P = dg::pullback( dg::coo3, g);
    for( unsigned i=0; i<g2d.size(); i++)
    {
        X[i] = g2d.map()[0][i];
        Y[i] = g2d.map()[1][i];
    }

    dg::HVec temp0( g2d.size()), temp1(temp0);
    dg::HVec w2d = dg::create::weights( g2d);

    err = nc_put_var_double( ncid, coordsID[0], periodify(X, g2d_periodic).data());
    err = nc_put_var_double( ncid, coordsID[1], periodify(Y, g2d_periodic).data());
    //err = nc_put_var_double( ncid, coordsID[2], g.z().data());

    //compute and write deformation into netcdf
    dg::SparseTensor<dg::HVec> metric = g2d.metric();
    dg::HVec g_xx = metric.value(0,0), g_yy=metric.value(1,1);
    dg::blas1::pointwiseDivide( g_yy, g_xx, temp0);
    const dg::HVec ones = dg::evaluate( dg::one, g2d);
    X=temp0;
    err = nc_put_var_double( ncid, defID, periodify(X, g2d_periodic).data());
    //compute and write conformalratio into netcdf
    dg::blas1::pointwiseDivide( g_yy, g_xx, temp0);
    X=temp0;
    err = nc_put_var_double( ncid, confID, periodify(X, g2d_periodic).data());

    std::cout << "Construction successful!\n";

    //compare determinant vs volume form
    dg::blas1::pointwiseDot( g_xx, g_yy, temp0);
    dg::blas1::axpby( 1., temp0, -1., temp1, temp0);
    dg::blas1::transform( temp0, temp0, dg::SQRT<double>());
    dg::blas1::pointwiseDivide( ones, temp0, temp0);
    dg::blas1::transfer( temp0, X);
    err = nc_put_var_double( ncid, volID, periodify(X, g2d_periodic).data());
    dg::SparseElement<dg::HVec> vol = dg::tensor::volume(metric);
    dg::blas1::axpby( 1., temp0, -1., vol.value(), temp0);
    double error = sqrt(dg::blas2::dot( temp0, w2d, temp0)/dg::blas2::dot( vol.value(), w2d, vol.value()));
    std::cout << "Rel Consistency  of volume is "<<error<<"\n";

    std::cout << "TEST VOLUME IS:\n";
    dg::HVec volume = dg::create::volume( g2d);
    dg::HVec ones2d = dg::evaluate( dg::one, g2d);
    double volumeUV = dg::blas1::dot( vol.value(), ones2d);

    volume = dg::create::volume( hector->internal_grid());
    ones2d = dg::evaluate( dg::one, hector->internal_grid());
    double volumeZE = dg::blas1::dot( vol.value(), ones2d);
    std::cout << "volumeUV is "<< volumeUV<<std::endl;
    std::cout << "volumeZE is "<< volumeZE<<std::endl;
    std::cout << "relative difference in volume is "<<fabs(volumeUV - volumeZE)/volumeZE<<std::endl;
    err = nc_close( ncid);
    delete hector;
    return 0;
}
