#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <fstream>
#include <sstream>
#include <cmath>

#include <mpi.h>

#include "dg/backend/xspacelib.cuh"
#include "dg/functors.h"

#include "dg/backend/timer.cuh"
#include "dg/backend/mpi_init.h"
#include "mpi_curvilinear.h"
//#include "guenther.h"
#include "solovev.h"
#include "ribeiro.h"
#include "simple_orthogonal.h"
//#include "ds.h"
#include "init.h"

#include <netcdf_par.h>
#include "file/nc_utilities.h"

using namespace dg::geo::solovev;
double sineX( double x, double y) {return sin(x)*sin(y);}
double cosineX( double x, double y) {return cos(x)*sin(y);}
double sineY( double x, double y) {return sin(x)*sin(y);}
double cosineY( double x, double y) {return sin(x)*cos(y);}

//should be the same as conformal_t.cu, except for the periodify
int main( int argc, char* argv[])
{
    MPI_Init( &argc, &argv);
    int rank;
    unsigned n, Nx, Ny, Nz; 
    MPI_Comm comm;
    dg::mpi_init3d( dg::DIR, dg::PER, dg::PER, n, Nx, Ny, Nz, comm);
    MPI_Comm_rank( MPI_COMM_WORLD, &rank);
    Json::Reader reader;
    Json::Value js;
    if( argc==1)
    {
        std::ifstream is("geometry_params_Xpoint.js");
        reader.parse(is,js,false);
    }
    else
    {
        std::ifstream is(argv[1]);
        reader.parse(is,js,false);
    }
    GeomParameters gp(js);
    dg::geo::BinaryFunctorsLvl2 psip = createPsip( gp);
    if(rank==0)std::cout << "Psi min "<<psip.f()(gp.R_0, 0)<<"\n";
    if(rank==0)std::cout << "Type psi_0 and psi_1\n";
    double psi_0, psi_1;
    if(rank==0)std::cin >> psi_0>> psi_1;
    MPI_Bcast( &psi_0, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
    MPI_Bcast( &psi_1, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
    if(rank==0)gp.display( std::cout);
    dg::Timer t;
    //solovev::detail::Fpsi fpsi( gp, -10);
    if(rank==0)std::cout << "Constructing grid ... \n";
    t.tic();
    dg::geo::Ribeiro ribeiro( psip, psi_0, psi_1, gp.R_0, 0., 1);
    dg::CurvilinearProductMPIGrid3d g3d(ribeiro, n, Nx, Ny,Nz, dg::DIR,dg::PER, dg::PER,comm);
    dg::CurvilinearMPIGrid2d g2d = g3d.perp_grid();
    t.toc();
    if(rank==0)std::cout << "Construction took "<<t.diff()<<"s"<<std::endl;
    int ncid;
    file::NC_Error_Handle err;
    MPI_Info info = MPI_INFO_NULL;
    err = nc_create_par( "test_mpi.nc", NC_NETCDF4|NC_MPIIO|NC_CLOBBER, g2d.communicator(), info, &ncid); //MPI ON
    int dim3d[2];
    err = file::define_dimensions(  ncid, dim3d, g2d.global());
    int coordsID[2], onesID, defID,confID, volID, divBID;
    err = nc_def_var( ncid, "x_XYP", NC_DOUBLE, 2, dim3d, &coordsID[0]);
    err = nc_def_var( ncid, "y_XYP", NC_DOUBLE, 2, dim3d, &coordsID[1]);
    //err = nc_def_var( ncid, "z_XYP", NC_DOUBLE, 3, dim3d, &coordsID[2]);
    err = nc_def_var( ncid, "psi", NC_DOUBLE, 2, dim3d, &onesID);
    err = nc_def_var( ncid, "deformation", NC_DOUBLE, 2, dim3d, &defID);
    err = nc_def_var( ncid, "conformal", NC_DOUBLE, 2, dim3d, &confID);
    err = nc_def_var( ncid, "volume", NC_DOUBLE, 2, dim3d, &volID);
    err = nc_def_var( ncid, "divB", NC_DOUBLE, 2, dim3d, &divBID);

    int dims[2], periods[2],  coords[2];
    MPI_Cart_get( g2d.communicator(), 2, dims, periods, coords);
    size_t count[2] = {g2d.n()*g2d.Ny(), g2d.n()*g2d.Nx()};
    size_t start[2] = {coords[1]*count[0], coords[0]*count[1]};
    err = nc_var_par_access( ncid, coordsID[0], NC_COLLECTIVE);
    err = nc_var_par_access( ncid, coordsID[1], NC_COLLECTIVE);
    err = nc_var_par_access( ncid, onesID, NC_COLLECTIVE);
    err = nc_var_par_access( ncid, defID, NC_COLLECTIVE);
    err = nc_var_par_access( ncid, divBID, NC_COLLECTIVE);

    dg::MHVec psi_p = dg::pullback( psip.f(), g2d);
    //g.display();
    err = nc_put_vara_double( ncid, onesID, start, count, psi_p.data().data());
    dg::HVec X( g2d.size()), Y(X); //P = dg::pullback( dg::coo3, g);
    for( unsigned i=0; i<g2d.size(); i++)
    {
        X[i] = g2d.map()[0].data()[i];
        Y[i] = g2d.map()[0].data()[i];
    }

    dg::MHVec temp0( dg::evaluate(dg::zero, g2d)), temp1(temp0);
    dg::MHVec w2d = dg::create::weights( g2d);

    err = nc_put_vara_double( ncid, coordsID[0], start,count, X.data());
    err = nc_put_vara_double( ncid, coordsID[1], start,count, Y.data());

    dg::SparseTensor<dg::MHVec> metric = g2d.metric();
    dg::MHVec g_xx = metric.value(0,0), g_xy = metric.value(0,1), g_yy=metric.value(1,1);
    dg::SparseElement<dg::MHVec> vol_ = dg::tensor::volume(metric);
    dg::MHVec vol = vol_.value();
    //err = nc_put_vara_double( ncid, coordsID[2], g.z().data());
    //dg::blas1::pointwiseDivide( g2d.g_xy(), g2d.g_xx(), temp0);
    dg::blas1::pointwiseDivide( g_yy, g_xx, temp0);
    const dg::MHVec ones = dg::evaluate( dg::one, g2d);
    dg::blas1::axpby( 1., ones, -1., temp0, temp0);
    dg::blas1::transfer( temp0.data(), X);
    err = nc_put_vara_double( ncid, defID, start,count, X.data());

    if(rank==0)std::cout << "Construction successful!\n";

    //compute error in volume element
    dg::blas1::pointwiseDot( g_xx, g_yy, temp0);
    dg::blas1::pointwiseDot( g_xy, g_xy, temp1);
    dg::blas1::axpby( 1., temp0, -1., temp1, temp0);
    dg::blas1::transfer( g_xx,  temp1);
    dg::blas1::pointwiseDot( temp1, temp1, temp1);
    dg::blas1::axpby( 1., temp1, -1., temp0, temp0);
    double error = sqrt( dg::blas2::dot( temp0, w2d, temp0)/dg::blas2::dot( temp1, w2d, temp1));
    if(rank==0)std::cout<< "Rel Error in Determinant is "<<error<<"\n";

    //compute error in determinant vs volume form
    dg::blas1::pointwiseDot( g_xx, g_yy, temp0);
    dg::blas1::pointwiseDot( g_xy, g_xy, temp1);
    dg::blas1::axpby( 1., temp0, -1., temp1, temp0);
    dg::blas1::transform( temp0, temp0, dg::SQRT<double>());
    dg::blas1::pointwiseDivide( ones, temp0, temp0);
    dg::blas1::transfer( temp0.data(), X);
    err = nc_put_var_double( ncid, volID, X.data());
    dg::blas1::axpby( 1., temp0, -1., vol, temp0);
    error = sqrt(dg::blas2::dot( temp0, w2d, temp0)/dg::blas2::dot( vol, w2d, vol));
    if(rank==0)std::cout << "Rel Consistency  of volume is "<<error<<"\n";

    //compare g^xx to volume form
    dg::blas1::transfer( g_xx, temp0);
    dg::blas1::pointwiseDivide( ones, temp0, temp0);
    dg::blas1::axpby( 1., temp0, -1., vol, temp0);
    error=sqrt(dg::blas2::dot( temp0, w2d, temp0))/sqrt( dg::blas2::dot(vol, w2d, vol));
    if(rank==0)std::cout << "Rel Error of volume form is "<<error<<"\n";

    vol = dg::create::volume( g3d);
    dg::MHVec ones3d = dg::evaluate( dg::one, g3d);
    double volume = dg::blas1::dot( vol, ones3d);

    if(rank==0)std::cout << "TEST VOLUME IS:\n";
    if( psi_0 < psi_1) gp.psipmax = psi_1, gp.psipmin = psi_0;
    else               gp.psipmax = psi_0, gp.psipmin = psi_1;
    dg::geo::Iris iris(psip.f(), gp.psipmin, gp.psipmax);
    //dg::CylindricalGrid3d<dg::HVec> g3d( gp.R_0 -2.*gp.a, gp.R_0 + 2*gp.a, -2*gp.a, 2*gp.a, 0, 2*M_PI, 3, 2200, 2200, 1, dg::PER, dg::PER, dg::PER);
    dg::CartesianMPIGrid2d g2dC( gp.R_0 -2.*gp.a, gp.R_0 + 2.*gp.a, -2.*gp.a, 2.*gp.a, 1, 2e3, 2e3, dg::DIR, dg::PER, g2d.communicator());
    dg::MHVec vec  = dg::evaluate( iris, g2dC);
    dg::MHVec R  = dg::evaluate( dg::cooX2d, g2dC);
    dg::MHVec g2d_weights = dg::create::volume( g2dC);
    double volumeRZP = 2.*M_PI*dg::blas2::dot( vec, g2d_weights, R);
    if(rank==0)std::cout << "volumeXYP is "<< volume<<std::endl;
    if(rank==0)std::cout << "volumeRZP is "<< volumeRZP<<std::endl;
    if(rank==0)std::cout << "relative difference in volume is "<<fabs(volumeRZP - volume)/volume<<std::endl;
    if(rank==0)std::cout << "Note that the error might also come from the volume in RZP!\n"; //since integration of jacobian is fairly good probably

    /////////////////////////TEST 3d grid//////////////////////////////////////
    //if(rank==0)std::cout << "Start DS test!"<<std::endl;
    //const dg::MHVec vol3d = dg::create::volume( g3d);
    //t.tic();
    ////DFA fieldaligned( CurvilinearField( gp, g3d.x(), g3d.f_x()), g3d, gp.rk4eps, dg::NoLimiter()); 
    //DFA fieldaligned( OrthogonalField( gp, g2d.global(), g2d.f2_xy()), g3d, gp.rk4eps, dg::NoLimiter()); 

    ////dg::DS<DFA, dg::MHMatrix, dg::MHVec> ds( fieldaligned, CurvilinearField(gp, g3d.x(), g3d.f_x()), dg::normed, dg::centered);
    //dg::DS<DFA, dg::MHMatrix, dg::MHVec> ds( fieldaligned, OrthogonalField(gp, g2d.global(), g2d.f2_xy()), dg::normed, dg::centered);
    //t.toc();
    //if(rank==0)std::cout << "Construction took "<<t.diff()<<"s\n";
    //dg::MHVec B = dg::pullback( dg::geo::InvB(gp), g3d), divB(B);
    //dg::MHVec lnB = dg::pullback( dg::geo::LnB(gp), g3d), gradB(B); //dg::MHVec gradLnB = dg::pullback( dg::geo::GradLnB(gp), g3d);
    //dg::blas1::pointwiseDivide( ones3d, B, B);
    //dg::MHVec function = dg::pullback( dg::geo::FuncNeu(gp), g3d), derivative(function);
    //ds( function, derivative);

    //ds.centeredT( B, divB);
    //double norm =  sqrt( dg::blas2::dot(divB, vol3d, divB));
    //if(rank==0)std::cout << "Divergence of B is "<<norm<<"\n";

    //ds.centered( lnB, gradB);
    //norm = sqrt(dg::blas2::dot(gradB,vol3d,gradB) );
    //if(rank==0)std::cout << "num. norm of gradLnB is "<<norm<<"\n";
    //norm = sqrt( dg::blas2::dot( gradLnB, vol3d, gradLnB) );
    //if(rank==0)std::cout << "ana. norm of gradLnB is "<<norm<<"\n";
    //dg::blas1::axpby( 1., gradB, -1., gradLnB, gradLnB);
    //X = divB.data();
    //err = nc_put_vara_double( ncid, divBID, start,count, X.data());
    //double norm2 = sqrt(dg::blas2::dot(gradLnB, vol3d,gradLnB));
    //if(rank==0)std::cout << "rel. error of lnB is    "<<norm2/norm<<"\n";
    err = nc_close( ncid);
    MPI_Finalize();


    return 0;
}
