#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <fstream>
#include <sstream>
#include <cmath>

#include "dg/backend/xspacelib.cuh"
#include "dg/functors.h"

#include "dg/backend/timer.cuh"
#include "curvilinear.h"
//#include "guenther.h"
#include "solovev.h"
#include "ribeiro.h"
//#include "ds.h"
#include "init.h"

#include "file/nc_utilities.h"

using namespace dg::geo::solovev;
thrust::host_vector<double> periodify( const thrust::host_vector<double>& in, const dg::Grid2d& g)
{
    thrust::host_vector<double> out(g.size());
    for( unsigned i=0; i<g.Ny()-1; i++)
    for( unsigned k=0; k<g.n(); k++)
    for( unsigned j=0; j<g.Nx(); j++)
    for( unsigned l=0; l<g.n(); l++)
        out[((i*g.n() + k)*g.Nx() + j)*g.n()+l] = 
            in[((i*g.n() + k)*g.Nx() + j)*g.n()+l];
    for( unsigned i=g.Ny()-1; i<g.Ny(); i++)
    for( unsigned k=0; k<g.n(); k++)
    for( unsigned j=0; j<g.Nx(); j++)
    for( unsigned l=0; l<g.n(); l++)
        out[((i*g.n() + k)*g.Nx() + j)*g.n()+l] = 
            in[((0*g.n() + k)*g.Nx() + j)*g.n()+l];
    return out;
}

double sineX( double x, double y) {return sin(x)*sin(y);}
double cosineX( double x, double y) {return cos(x)*sin(y);}
double sineY( double x, double y) {return sin(x)*sin(y);}
double cosineY( double x, double y) {return sin(x)*cos(y);}

int main( int argc, char* argv[])
{
    std::cout << "Type n, Nx, Ny, Nz\n";
    unsigned n, Nx, Ny, Nz;
    std::cin >> n>> Nx>>Ny>>Nz;   
    Json::Reader reader;
    Json::Value js;
    if( argc==1)
    {
        std::ifstream is("geometry_params_Xpoint.js");
        reader.parse(is,js,false);
    }
    else
    {
        std::ifstream is(argv[1]);
        reader.parse(is,js,false);
    }
    //write parameters from file into variables
    dg::geo::solovev::GeomParameters gp(js);
    dg::geo::BinaryFunctorsLvl2 psip = dg::geo::solovev::createPsip( gp);
    std::cout << "Psi min "<<psip.f()(gp.R_0, 0)<<"\n";
    std::cout << "Type psi_0 and psi_1\n";
    double psi_0, psi_1;
    std::cin >> psi_0>> psi_1;
    gp.display( std::cout);
    dg::Timer t;
    //solovev::detail::Fpsi fpsi( gp, -10);
    std::cout << "Constructing ribeiro grid ... \n";
    t.tic();
    dg::geo::Ribeiro ribeiro( psip, psi_0, psi_1, gp.R_0, 0., 1);
    dg::CurvilinearProductGrid3d g3d(ribeiro, n, Nx, Ny,Nz, dg::DIR);
    dg::CurvilinearGrid2d g2d = g3d.perp_grid();
    dg::Grid2d g2d_periodic(g2d.x0(), g2d.x1(), g2d.y0(), g2d.y1(), g2d.n(), g2d.Nx(), g2d.Ny()+1); 
    t.toc();
    std::cout << "Construction took "<<t.diff()<<"s"<<std::endl;
    int ncid;
    file::NC_Error_Handle err;
    err = nc_create( "ribeiro.nc", NC_NETCDF4|NC_CLOBBER, &ncid);
    int dim3d[2];
    err = file::define_dimensions(  ncid, dim3d, g2d_periodic);
    int coordsID[2], onesID, defID, confID, volID,divBID;
    err = nc_def_var( ncid, "x_XYP", NC_DOUBLE, 2, dim3d, &coordsID[0]);
    err = nc_def_var( ncid, "y_XYP", NC_DOUBLE, 2, dim3d, &coordsID[1]);
    //err = nc_def_var( ncid, "z_XYP", NC_DOUBLE, 3, dim3d, &coordsID[2]);
    err = nc_def_var( ncid, "psi", NC_DOUBLE, 2, dim3d, &onesID);
    err = nc_def_var( ncid, "deformation", NC_DOUBLE, 2, dim3d, &defID);
    err = nc_def_var( ncid, "conformal", NC_DOUBLE, 2, dim3d, &confID);
    err = nc_def_var( ncid, "volume", NC_DOUBLE, 2, dim3d, &volID);
    err = nc_def_var( ncid, "divB", NC_DOUBLE, 2, dim3d, &divBID);

    thrust::host_vector<double> psi_p = dg::pullback( psip.f(), g2d);
    //g.display();
    err = nc_put_var_double( ncid, onesID, periodify(psi_p, g2d_periodic).data());
    dg::HVec X( g2d.size()), Y(X); //P = dg::pullback( dg::coo3, g);
    for( unsigned i=0; i<g2d.size(); i++)
    {
        X[i] = g2d.map()[0][i];
        Y[i] = g2d.map()[1][i];
    }

    dg::HVec temp0( g2d.size()), temp1(temp0);
    dg::HVec w2d = dg::create::weights( g2d);

    err = nc_put_var_double( ncid, coordsID[0], periodify(X, g2d_periodic).data());
    err = nc_put_var_double( ncid, coordsID[1], periodify(Y, g2d_periodic).data());

    dg::SparseTensor<dg::HVec> metric = g2d.metric();
    dg::HVec g_xx = metric.value(0,0), g_xy = metric.value(0,1), g_yy=metric.value(1,1);
    dg::SparseElement<dg::HVec> vol_ = dg::tensor::volume(metric);
    dg::HVec vol = vol_.value();
    //err = nc_put_var_double( ncid, coordsID[2], g.z().data());
    //compute and write deformation into netcdf
    dg::blas1::pointwiseDivide( g_xy, g_xx, temp0);
    const dg::HVec ones = dg::evaluate( dg::one, g2d);
    X=g_yy;
    err = nc_put_var_double( ncid, defID, periodify(X, g2d_periodic).data());
    //compute and write ribeiroratio into netcdf
    dg::blas1::pointwiseDivide( g_yy, g_xx, temp0);
    X=temp0;

    err = nc_put_var_double( ncid, confID, periodify(X, g2d_periodic).data());
    std::cout << "Construction successful!\n";

    //compute error in volume element (in ribeiro grid g^xx is the volume element)
    dg::blas1::pointwiseDot( g_xx, g_yy, temp0);
    dg::blas1::pointwiseDot( g_xy, g_xy, temp1);
    dg::blas1::axpby( 1., temp0, -1., temp1, temp0);
    dg::blas1::transfer( g_xx,  temp1);
    dg::blas1::pointwiseDot( temp1, temp1, temp1);
    dg::blas1::axpby( 1., temp1, -1., temp0, temp0);
    double error = sqrt( dg::blas2::dot( temp0, w2d, temp0)/dg::blas2::dot( temp1, w2d, temp1));
    std::cout<< "Rel Error in Determinant is "<<error<<"\n";

    //compute error in determinant vs volume form
    dg::blas1::pointwiseDot( g_xx, g_yy, temp0);
    dg::blas1::pointwiseDot( g_xy, g_xy, temp1);
    dg::blas1::axpby( 1., temp0, -1., temp1, temp0);
    dg::blas1::transform( temp0, temp0, dg::SQRT<double>());
    dg::blas1::pointwiseDivide( ones, temp0, temp0);
    dg::blas1::transfer( temp0, X);
    err = nc_put_var_double( ncid, volID, periodify(X, g2d_periodic).data());
    dg::blas1::axpby( 1., temp0, -1., vol, temp0);
    error = sqrt(dg::blas2::dot( temp0, w2d, temp0)/dg::blas2::dot( vol, w2d, vol));
    std::cout << "Rel Consistency  of volume is "<<error<<"\n";

    //compare g^xx to volume form
    dg::blas1::transfer( g_xx, temp0);
    dg::blas1::pointwiseDivide( ones, temp0, temp0);
    dg::blas1::axpby( 1., temp0, -1., vol, temp0);
    error=sqrt(dg::blas2::dot( temp0, w2d, temp0))/sqrt( dg::blas2::dot(vol, w2d, vol));
    std::cout << "Rel Error of volume form is "<<error<<"\n";

    vol = dg::create::volume( g3d);
    dg::HVec ones3d = dg::evaluate( dg::one, g3d);
    double volume = dg::blas1::dot( vol, ones3d);

    std::cout << "TEST VOLUME IS:\n";
    if( psi_0 < psi_1) gp.psipmax = psi_1, gp.psipmin = psi_0;
    else               gp.psipmax = psi_0, gp.psipmin = psi_1;
    dg::geo::Iris iris(psip.f(), gp.psipmin, gp.psipmax);
    //dg::CylindricalGrid3d<dg::HVec> g3d( gp.R_0 -2.*gp.a, gp.R_0 + 2*gp.a, -2*gp.a, 2*gp.a, 0, 2*M_PI, 3, 2200, 2200, 1, dg::PER, dg::PER, dg::PER);
//     dg::CartesianGrid2d g2dC( gp.R_0 -1.2*gp.a, gp.R_0 + 1.2*gp.a, -1.2*gp.a, 1.2*gp.a, 1, 1e3, 1e3, dg::PER, dg::PER);
    dg::CartesianGrid2d g2dC( gp.R_0 -2.0*gp.a, gp.R_0 + 2.0*gp.a, -2.0*gp.a, 2.0*gp.a, 1, 2e3, 2e3, dg::PER, dg::PER);
    dg::HVec vec  = dg::evaluate( iris, g2dC);
    dg::HVec R  = dg::evaluate( dg::cooX2d, g2dC);
    dg::HVec g2d_weights = dg::create::volume( g2dC);
    double volumeRZP = 2.*M_PI*dg::blas2::dot( vec, g2d_weights, R);
    std::cout << "volumeXYP is "<< volume<<std::endl;
    std::cout << "volumeRZP is "<< volumeRZP<<std::endl;
    std::cout << "relative difference in volume is "<<fabs(volumeRZP - volume)/volume<<std::endl;
    std::cout << "Note that the error might also come from the volume in RZP!\n"; //since integration of jacobian is fairly good probably

    /////////////////////////TEST 3d grid//////////////////////////////////////
    //std::cout << "Start DS test!"<<std::endl;
    //const dg::HVec vol3d = dg::create::volume( g3d);
    //t.tic();
    //DFA fieldaligned( dg::ribeiro::Field( gp, g3d.x(), g3d.f_x()), g3d, gp.rk4eps, dg::NoLimiter()); 

    //dg::DS<DFA, dg::DMatrix, dg::HVec> ds( fieldaligned, dg::ribeiro::Field(gp, g3d.x(), g3d.f_x()), dg::normed, dg::centered);

    //t.toc();
    //std::cout << "Construction took "<<t.diff()<<"s\n";
    //dg::HVec B = dg::pullback( dg::geo::InvB(gp), g3d), divB(B);
    //dg::HVec lnB = dg::pullback( dg::geo::LnB(gp), g3d), gradB(B);
    //dg::HVec gradLnB = dg::pullback( dg::geo::GradLnB(gp), g3d);
    //dg::blas1::pointwiseDivide( ones3d, B, B);
    //dg::HVec function = dg::pullback( dg::geo::FuncNeu(gp), g3d), derivative(function);
    //ds( function, derivative);

    //ds.centeredT( B, divB);
    //double norm =  sqrt( dg::blas2::dot(divB, vol3d, divB));
    //std::cout << "Divergence of B is "<<norm<<"\n";

    //ds.centered( lnB, gradB);
    //std::cout << "num. norm of gradLnB is "<<sqrt( dg::blas2::dot( gradB,vol3d, gradB))<<"\n";
    //norm = sqrt( dg::blas2::dot( gradLnB, vol3d, gradLnB) );
    //std::cout << "ana. norm of gradLnB is "<<norm<<"\n";
    //dg::blas1::axpby( 1., gradB, -1., gradLnB, gradLnB);
    //X = divB;
    //err = nc_put_var_double( ncid, divBID, periodify(X, g2d_periodic).data());
    //double norm2 = sqrt(dg::blas2::dot(gradLnB, vol3d,gradLnB));
    //std::cout << "rel. error of lnB is    "<<norm2/norm<<"\n";
    //err = nc_close( ncid);



    return 0;
}
