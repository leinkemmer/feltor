#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <fstream>
#include <sstream>
#include <cmath>

#include "dg/backend/xspacelib.cuh"
#include "dg/functors.h"

#include "dg/backend/timer.cuh"
#include "solovev.h"
#include "taylor.h"
//#include "guenther.h"
#include "dg/geometry/transform.h"
#include "refined_curvilinearX.h"
#include "curvilinearX.h"
#include "separatrix_orthogonal.h"
#include "init.h"

#include "file/nc_utilities.h"


struct ZCutter
{
    ZCutter(double ZX): Z_X(ZX){}
    double operator()(double R, double Z) const {
        if( Z> Z_X) 
            return 1;
        return 0;
    }
    private:
    double Z_X;
};
double sine( double x) {return sin(x);}
double cosine( double x) {return cos(x);}
//typedef dg::FieldAligned< dg::CurvilinearGridX3d<dg::HVec> , dg::IHMatrix, dg::HVec> HFA;

thrust::host_vector<double> periodify( const thrust::host_vector<double>& in, const dg::GridX3d& g)
{
    assert( g.Nz() == 2);
    thrust::host_vector<double> out(g.size());
    for( unsigned s=0; s<g.Nz(); s++)
    for( unsigned i=0; i<g.Ny(); i++)
    for( unsigned k=0; k<g.n(); k++)
    for( unsigned j=0; j<g.Nx(); j++)
    for( unsigned l=0; l<g.n(); l++)
        out[(((s*g.Ny()+i)*g.n() + k)*g.Nx() + j)*g.n()+l] = 
            in[((i*g.n() + k)*g.Nx() + j)*g.n()+l];

    //exchange two segments
    for( unsigned i=g.outer_Ny(); i<2*g.outer_Ny(); i++)
    for( unsigned k=0; k<g.n(); k++)
    for( unsigned j=0; j<g.Nx(); j++)
    for( unsigned l=0; l<g.n(); l++)
        out[(((1*g.Ny() + i)*g.n() + k)*g.Nx() + j)*g.n()+l] = 
            in[(((i+g.inner_Ny())*g.n() + k)*g.Nx() + j)*g.n()+l];
    for( unsigned i=g.inner_Ny()+g.outer_Ny(); i<g.Ny(); i++)
    for( unsigned k=0; k<g.n(); k++)
    for( unsigned j=0; j<g.Nx(); j++)
    for( unsigned l=0; l<g.n(); l++)
        out[(((1*g.Ny() + i)*g.n() + k)*g.Nx() + j)*g.n()+l] = 
            in[(((i-g.inner_Ny())*g.n() + k)*g.Nx() + j)*g.n()+l];
    if( g.outer_Ny() == 0)
    {
    //exchange two segments
    for( unsigned i=0; i<g.Ny()-1; i++)
    for( unsigned k=0; k<g.n(); k++)
    for( unsigned j=0; j<g.Nx(); j++)
    for( unsigned l=0; l<g.n(); l++)
        out[(((1*g.Ny() + i)*g.n() + k)*g.Nx() + j)*g.n()+l] = 
            in[(((i+1)*g.n() + k)*g.Nx() + j)*g.n()+l];
    for( unsigned i=g.Ny()-1; i<g.Ny(); i++)
    for( unsigned k=0; k<g.n(); k++)
    for( unsigned j=0; j<g.Nx(); j++)
    for( unsigned l=0; l<g.n(); l++)
        out[(((1*g.Ny() + i)*g.n() + k)*g.Nx() + j)*g.n()+l] = 
            in[(((0)*g.n() + k)*g.Nx() + j)*g.n()+l];
    }


    return out;
}

int main( int argc, char* argv[])
{
    std::cout << "Type n, Nx, Ny, Nz \n";
    unsigned n, Nx, Ny, Nz;
    std::cin >> n>> Nx>>Ny>>Nz;   
    Json::Reader reader;
    Json::Value js;
    if( argc==1)
    {
        std::ifstream is("geometry_params_Xpoint_taylor.js");
        reader.parse(is,js,false);
    }
    else
    {
        std::ifstream is(argv[1]);
        reader.parse(is,js,false);
    }
    dg::geo::taylor::GeomParameters gp(js);
    dg::Timer t;
    std::cout << "Type psi_0 \n";
    double psi_0 = -16;
    std::cin >> psi_0;
    std::cout << "Type fx and fy ( fx*Nx and fy*Ny must be integer) \n";
    double fx_0=1./8., fy_0=1./22.;
    std::cin >> fx_0>> fy_0;
    std::cout << "Type add_x and add_y \n";
    double add_x, add_y;
    std::cin >> add_x >> add_y;
    gp.display( std::cout);
    std::cout << "Constructing orthogonal grid ... \n";
    t.tic();
    dg::geo::TokamakMagneticField c = dg::geo::createTaylorField(gp);
    std::cout << "Psi min "<<c.psip()(gp.R_0, 0)<<"\n";
    double R_X = gp.R_0-1.1*gp.triangularity*gp.a;
    double Z_X = -1.1*gp.elongation*gp.a;
    dg::geo::findXpoint( c.get_psip(), R_X, Z_X);

    double R0 = gp.R_0, Z0 = 0;
    dg::geo::SeparatrixOrthogonal generator(c.get_psip(), psi_0, R_X,Z_X, R0, Z0,0);
    //dg::geo::SimpleOrthogonalX generator(c.get_psip(), psi_0, R_X,Z_X, R0, Z0,0);
    dg::EquidistXRefinement equi(add_x, add_y, 1,1);
    dg::CurvilinearRefinedProductGridX3d g3d(equi, generator, fx_0, fy_0, n, Nx, Ny,Nz, dg::DIR, dg::NEU);
    dg::CurvilinearRefinedGridX2d g2d(equi, generator, fx_0, fy_0, n, Nx, Ny,dg::DIR, dg::NEU);
    t.toc();
    dg::GridX3d g3d_periodic(g3d.x0(), g3d.x1(), g3d.y0(), g3d.y1(), g3d.z0(), g3d.z1(), g3d.fx(), g3d.fy(), g3d.n(), g3d.Nx(), g3d.Ny(), 2); 
    std::cout << "Construction took "<<t.diff()<<"s"<<std::endl;
    double psi_1 = -fx_0/(1.-fx_0)*psi_0;
    std::cout << "psi 1 is          "<<psi_1<<"\n";
    dg::Grid1d g1d( g2d.x0(), g2d.x1(), g2d.n(), g2d.Nx());
    g1d.display( std::cout);
    dg::HVec x_left = dg::evaluate( sine, g1d), x_right(x_left);
    dg::HVec y_left = dg::evaluate( cosine, g1d);
    int ncid;
    file::NC_Error_Handle err;
    err = nc_create( "orthogonalX.nc", NC_NETCDF4|NC_CLOBBER, &ncid);
    int dim3d[3], dim1d[1];
    err = file::define_dimensions(  ncid, dim3d, g3d_periodic.grid());
    //err = file::define_dimensions(  ncid, dim3d, g2d.grid());
    err = file::define_dimension(  ncid, "i", dim1d, g1d);
    int coordsID[2], onesID, defID, volID, divBID;
    int coord1D[5];
    err = nc_def_var( ncid, "x_XYP", NC_DOUBLE, 3, dim3d, &coordsID[0]);
    err = nc_def_var( ncid, "y_XYP", NC_DOUBLE, 3, dim3d, &coordsID[1]);
    err = nc_def_var( ncid, "x_left", NC_DOUBLE, 1, dim1d, &coord1D[0]);
    err = nc_def_var( ncid, "y_left", NC_DOUBLE, 1, dim1d, &coord1D[1]);
    err = nc_def_var( ncid, "x_right", NC_DOUBLE, 1, dim1d, &coord1D[2]);
    err = nc_def_var( ncid, "y_right", NC_DOUBLE, 1, dim1d, &coord1D[3]);
    err = nc_def_var( ncid, "f_x", NC_DOUBLE, 1, dim1d, &coord1D[4]);
    //err = nc_def_var( ncid, "z_XYP", NC_DOUBLE, 3, dim3d, &coordsID[2]);
    err = nc_def_var( ncid, "psi", NC_DOUBLE, 3, dim3d, &onesID);
    err = nc_def_var( ncid, "deformation", NC_DOUBLE, 3, dim3d, &defID);
    err = nc_def_var( ncid, "volume", NC_DOUBLE, 3, dim3d, &volID);
    err = nc_def_var( ncid, "divB", NC_DOUBLE, 3, dim3d, &divBID);

    thrust::host_vector<double> psi_p = dg::pullback( c.psip(), g2d);
    g2d.display();
    err = nc_put_var_double( ncid, onesID, periodify(psi_p, g3d_periodic).data());
    //err = nc_put_var_double( ncid, onesID, periodify(g2d.g(), g3d_periodic).data());
    dg::HVec X( g2d.size()), Y(X); //P = dg::pullback( dg::coo3, g);
    for( unsigned i=0; i<g2d.size(); i++)
    {
        X[i] = g2d.map()[0][i];
        Y[i] = g2d.map()[1][i];
    }

    dg::HVec ones = dg::evaluate( dg::one, g2d);
    dg::HVec temp0( g2d.size()), temp1(temp0);
    dg::HVec w2d = dg::create::weights( g2d);

    err = nc_put_var_double( ncid, coordsID[0], periodify(X, g3d_periodic).data());
    err = nc_put_var_double( ncid, coordsID[1], periodify(Y, g3d_periodic).data());
    //err = nc_put_var_double( ncid, coordsID[0], X.data());
    //err = nc_put_var_double( ncid, coordsID[1], Y.data());
    //err = nc_put_var_double( ncid, coord1D[0], g3d.rx0().data());
    //err = nc_put_var_double( ncid, coord1D[1], g3d.zx0().data());
    //err = nc_put_var_double( ncid, coord1D[2], g3d.rx1().data());
    //err = nc_put_var_double( ncid, coord1D[3], g3d.zx1().data());
    //err = nc_put_var_double( ncid, coord1D[4], periodify(g3d.f_x(), g3d_periodic).data());
    //err = nc_put_var_double( ncid, coord1D[4], g3d.f_x().data());
    //err = nc_put_var_double( ncid, coordsID[2], g.z().data());

    dg::SparseTensor<dg::HVec> metric = g2d.metric();
    dg::HVec g_xx = metric.value(0,0), g_yy=metric.value(1,1);
    dg::SparseElement<dg::HVec> vol_ = dg::tensor::volume(metric);
    dg::HVec vol = vol_.value();
    dg::blas1::pointwiseDivide( g_yy, g_xx, temp0);
    dg::blas1::axpby( 1., ones, -1., temp0, temp0);
    dg::blas1::transfer( temp0, X);
    err = nc_put_var_double( ncid, defID, periodify(X, g3d_periodic).data());
    //err = nc_put_var_double( ncid, defID, X.data());
    dg::blas1::transfer( vol, X);
    dg::blas1::transfer( g_yy, Y);
    dg::blas1::pointwiseDot( Y, X, X);
    err = nc_put_var_double( ncid, volID, periodify(X, g3d_periodic).data());
    //err = nc_put_var_double( ncid, volID, X.data());

    std::cout << "Construction successful!\n";

    //compare determinant vs volume form
    dg::blas1::pointwiseDot( g_xx, g_yy, temp0);
    dg::blas1::axpby( 1., temp0, -1., temp1, temp0);
    dg::blas1::transform( temp0, temp0, dg::SQRT<double>());
    dg::blas1::pointwiseDivide( ones, temp0, temp0);
    dg::blas1::transfer( temp0, X);
    err = nc_put_var_double( ncid, volID, periodify(X, g3d_periodic).data());
    dg::blas1::axpby( 1., temp0, -1., vol, temp0);
    double error = sqrt(dg::blas2::dot( temp0, w2d, temp0)/dg::blas2::dot( vol, w2d, vol));
    //error = sqrt(dg::blas1::dot( temp0, temp0)/dg::blas1::dot( g2d.vol(), g2d.vol()));
    std::cout << "Rel Consistency  of volume is "<<error<<"\n";

    ////alternative method to compute volume
    //solovev::PsipR psipR( gp);
    //solovev::PsipZ psipZ( gp);
    //dg::HVec psipR_ = dg::pullback(psipR, g2d);
    //dg::HVec psipZ_ = dg::pullback(psipZ, g2d);
    //dg::HVec psip2_(psipR_);
    //dg::blas1::pointwiseDot( psipR_, psipR_, psipR_);
    //dg::blas1::pointwiseDot( psipZ_, psipZ_, psipZ_);
    //dg::blas1::axpby( 1., psipR_, 1., psipZ_, psip2_);
    //const dg::HVec f_ = g2d.f();
    //const dg::HVec g_ = g2d.g();
    //dg::blas1::pointwiseDot( f_, f_, temp1);
    //dg::blas1::pointwiseDot( psip2_, temp1, temp1);
    //dg::blas1::pointwiseDot( g2d.weightsX(), temp1, temp1);
    //dg::blas1::pointwiseDot( g2d.weightsX(), temp1, temp1);
    //dg::blas1::axpby( 1., g2d.g_xx(), -1., temp1, temp1);
    //error= dg::blas2::dot( temp1, w2d, temp1)/dg::blas2::dot(g2d.g_xx(),w2d,g2d.g_xx());
    ////error= dg::blas1::dot( temp1, temp1)/dg::blas1::dot(g2d.g_xx(),g2d.g_xx());
    //std::cout << "Rel Error of g_xx is "<<sqrt(error)<<"\n";
    //dg::blas1::pointwiseDot( g_, g_, temp1);
    //dg::blas1::pointwiseDot( psip2_,  temp1, temp1);
    //dg::blas1::pointwiseDot( g2d.weightsY(), temp1, temp1);
    //dg::blas1::pointwiseDot( g2d.weightsY(), temp1, temp1);
    //dg::blas1::axpby( 1., g2d.g_yy(), -1., temp1, temp1);
    //error= dg::blas2::dot( temp1, w2d, temp1)/dg::blas2::dot(g2d.g_yy(),w2d,g2d.g_yy());
    ////error= dg::blas1::dot( temp1, temp1)/dg::blas1::dot(g2d.g_yy(),g2d.g_yy());
    //std::cout << "Rel Error of g_yy is "<<sqrt(error)<<"\n";
    //dg::blas1::pointwiseDivide( ones, g2d.vol(), temp0);
    //dg::blas1::pointwiseDot( f_, psip2_, temp1);
    //dg::blas1::pointwiseDot( g_, temp1 , temp1);
    //dg::blas1::pointwiseDot( g2d.weightsX(), temp1, temp1);
    //dg::blas1::pointwiseDot( g2d.weightsY(), temp1, temp1);
    //dg::blas1::axpby( 1., temp0, -1., temp1, temp1);
    //error= dg::blas2::dot( temp1, w2d, temp1)/dg::blas2::dot(temp0,w2d,temp0);
    ////error= dg::blas1::dot( temp1, temp1)/dg::blas1::dot(temp0,temp0);
    //std::cout << "Rel Error of volume is "<<sqrt(error)<<"\n";
    //
    err = nc_close( ncid);

    std::cout << "TEST VOLUME IS:\n";
    dg::CartesianGrid2d g2dC( gp.R_0 -1.2*gp.a, gp.R_0 + 1.2*gp.a, Z_X, 1.2*gp.a*gp.elongation, 1, 5e3, 5e3, dg::PER, dg::PER);
    gp.psipmax = 0., gp.psipmin = psi_0;
    dg::geo::Iris iris( c.psip(), gp.psipmin, gp.psipmax);
    dg::HVec vec  = dg::evaluate( iris, g2dC);
    dg::HVec g2d_weights = dg::create::volume( g2dC);
    double volumeRZP = dg::blas1::dot( vec, g2d_weights);

    dg::HVec cutter = dg::pullback( iris, g2d), vol_cut( cutter);
    ZCutter cut(Z_X);
    dg::HVec zcutter = dg::pullback( cut, g2d); 
    w2d = dg::create::weights( g2d);//make weights w/o refined weights
    dg::blas1::pointwiseDot(cutter, w2d, vol_cut);
    dg::blas1::pointwiseDot(zcutter, vol_cut, vol_cut);
    double volume = dg::blas1::dot( vol, vol_cut);
    std::cout << "volumeXYP is "<< volume<<std::endl;
    std::cout << "volumeRZP is "<< volumeRZP<<std::endl;
    std::cout << "relative difference in volume is "<<fabs(volumeRZP - volume)/volume<<std::endl;
    std::cout << "Note that the error might also be because the regions in the RZ grid and the orthogonal grid are not the same!\n";
    std::cout << "Note that the error might also come from the volume in RZP!\n";

   // ///////////////////////////TEST 3d grid//////////////////////////////////////
   // std::cout << "Start DS test!"<<std::endl;
   // const dg::HVec vol3d = dg::create::volume( g3d);
   // //DFA fieldaligned(OrthogonalXField( gp, g2d, g2d.g()), g3d, gp.rk4eps, dg::NoLimiter(), dg::NEU); 
   // DFA fieldaligned( ConformalField( gp, g2d.x(), g2d.f_x()), g3d, gp.rk4eps, dg::NoLimiter(), dg::NEU); 

   // //dg::DS<DFA, dg::Composite<dg::DMatrix>, dg::HVec> ds( fieldaligned, OrthogonalXField(gp, g2d, g2d.g()), dg::normed, dg::centered, false);
   // dg::DS<DFA, dg::Composite<dg::DMatrix>, dg::HVec> ds( fieldaligned, ConformalField(gp, g2d.x(), g2d.f_x()), dg::normed, dg::centered, false);
   // dg::HVec B = dg::pullback( solovev::InvB(gp), g3d), divB(B);
   // dg::HVec lnB = dg::pullback( solovev::LnB(gp), g3d), gradB(B);
   // const dg::HVec gradLnB = dg::pullback( solovev::GradLnB(gp), g3d);
   // dg::blas1::pointwiseDivide( ones, B, B);

   // ds.centeredT( B, divB);
   // std::cout << "Divergence of B is "<<sqrt( dg::blas2::dot( divB, vol3d, divB))<<"\n";
   // ds.centered( lnB, gradB);
   // dg::blas1::axpby( 1., gradB, -1., gradLnB, gradB);
   // //test if topological shift was correct!!
   // X = gradB;
   // dg::blas1::pointwiseDot(cutter, gradB, gradB);
   // double norm = sqrt( dg::blas2::dot( gradLnB, vol3d, gradLnB) );
   // std::cout << "rel. error of lnB is    "<<sqrt( dg::blas2::dot( gradB, vol3d, gradB))/norm<<" (doesn't fullfill boundary conditions so it was cut at separatrix)\n";

   // const dg::HVec function = dg::pullback(solovev::FuncNeu(gp), g3d);
   // dg::HVec temp(function);
   // const dg::HVec derivative = dg::pullback(solovev::DeriNeu(gp), g3d);
   // ds( function, temp);
   // dg::blas1::axpby( 1., temp, -1., derivative, temp);
   // norm = sqrt( dg::blas2::dot( derivative, vol3d, derivative) );
   // std::cout << "rel. error of DS  is    "<<sqrt( dg::blas2::dot( temp, vol3d, temp))/norm<<"\n";
   // err = nc_put_var_double( ncid, divBID, periodify(X, g3d_periodic).data());
   // //err = nc_put_var_double( ncid, divBID, X.data());
    //dg::DVec psiphom = dg::pullback( solovev::FuncDirNeu(gp, psi_0, g3d.psi1()), g2d);
    //dg::DVec psiphom = dg::pullback( solovev::mod::Psip(gp), g2d);
    //dg::blas1::transfer( g2d.g(), X);
    //dg::blas1::transfer( psiphom, X);
    //err = nc_put_var_double( ncid, divBID, periodify(X, g3d_periodic).data());


    return 0;
}
