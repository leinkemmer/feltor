#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <sstream>
#include <cmath>

#include <mpi.h> //activate mpi

#include "dg/algorithm.h"
#include "dg/backend/timer.cuh"
#include "dg/backend/xspacelib.cuh"
#include "dg/backend/interpolation.cuh"

#include "netcdf_par.h" //exclude if par netcdf=OFF
#include "file/nc_utilities.h"

#include "asela.cuh"

/*
    - the only difference to the asela_hpc.cu file is that this program 
        uses the MPI backend and
        the parallel netcdf output 
    - pay attention that both the grid dimensions as well as the 
        output dimensions must be divisible by the mpi process numbers
*/

typedef dg::MPI_FieldAligned< dg::CylindricalMPIGrid3d, dg::IDMatrix,dg::BijectiveComm< dg::iDVec, dg::DVec >, dg::DVec> DFA;
using namespace dg::geo::solovev;
int main( int argc, char* argv[])
{
    ////////////////////////////////setup MPI///////////////////////////////
    int provided;
    MPI_Init_thread( &argc, &argv, MPI_THREAD_FUNNELED, &provided);
    if( provided != MPI_THREAD_FUNNELED)
    {
        std::cerr << "wrong mpi-thread environment provided!\n";
        return -1;
    }
    int periods[3] = {false, false, true}; //non-, non-, periodic
    int rank, size;
    MPI_Comm_rank( MPI_COMM_WORLD, &rank);
    MPI_Comm_size( MPI_COMM_WORLD, &size);
#if THRUST_DEVICE_SYSTEM==THRUST_DEVICE_SYSTEM_CUDA
    int num_devices=0;
    hipGetDeviceCount(&num_devices);
    if(num_devices==0){std::cerr << "No CUDA capable devices found"<<std::endl; return -1;}
    int device = rank % num_devices; //assume # of gpus/node is fixed
    hipSetDevice( device);
#endif//cuda
    int np[3];
    if(rank==0)
    {
        std::cin>> np[0] >> np[1] >>np[2];
        std::cout << "Computing with "<<np[0]<<" x "<<np[1]<<" x "<<np[2] << " = "<<size<<std::endl;
        assert( size == np[0]*np[1]*np[2]);
    }
    MPI_Bcast( np, 3, MPI_INT, 0, MPI_COMM_WORLD);
    MPI_Comm comm;
    MPI_Cart_create( MPI_COMM_WORLD, 3, np, periods, true, &comm);
    ////////////////////////Parameter initialisation//////////////////////////
    Json::Reader reader;
    Json::Value js, gs;
    if( argc != 4)
    {
        if(rank==0)std::cerr << "ERROR: Wrong number of arguments!\nUsage: "<< argv[0]<<" [inputfile] [geomfile] [outputfile]\n";
        return -1;
    }
    else 
    {
        std::ifstream is(argv[1]);
        std::ifstream ks(argv[2]);
        reader.parse(is,js,false);
        reader.parse(ks,gs,false);
    }
    const eule::Parameters p( js);
    const dg::geo::solovev::GeomParameters gp(gs);
    if(rank==0)p.display( std::cout);
    if(rank==0)gp.display( std::cout);
    std::string input = js.toStyledString(), geom = gs.toStyledString();
    ////////////////////////////////set up computations///////////////////////////
    
    double Rmin=gp.R_0-p.boxscaleRm*gp.a;
    double Zmin=-p.boxscaleZm*gp.a*gp.elongation;
    double Rmax=gp.R_0+p.boxscaleRp*gp.a; 
    double Zmax=p.boxscaleZp*gp.a*gp.elongation;
    //Make grids
     dg::CylindricalMPIGrid3d grid( Rmin,Rmax, Zmin,Zmax, 0, 2.*M_PI, p.n, p.Nx, p.Ny, p.Nz, p.bc, p.bc, dg::PER, comm);  
     dg::CylindricalMPIGrid3d grid_out( Rmin,Rmax, Zmin,Zmax, 0, 2.*M_PI, p.n_out, p.Nx_out, p.Ny_out, p.Nz_out, p.bc, p.bc, dg::PER, comm);  

     
    //create RHS 
    if(rank==0)std::cout << "Constructing Asela...\n";
    eule::Asela<dg::CylindricalMPIGrid3d, dg::DS<DFA, dg::MDMatrix, dg::MDVec>, dg::MDMatrix, dg::MDVec> asela( grid, p, gp); //initialize before rolkar!
    if(rank==0)std::cout << "Constructing Rolkar...\n";
    eule::Rolkar< dg::CylindricalMPIGrid3d, dg::DS<DFA, dg::MDMatrix, dg::MDVec>, dg::MDMatrix, dg::MDVec > rolkar( grid, p, gp, asela.ds(), asela.dsDIR());
    if(rank==0)std::cout << "Done!\n";

    /////////////////////The initial field/////////////////////////////////////////
    //background profile
    dg::geo::Nprofile<Psip> prof(p.bgprofamp, p.nprofileamp, gp, Psip(gp)); //initial background profile
    std::vector<dg::MDVec> y0(4, dg::evaluate( prof, grid)), y1(y0); 
    //perturbation 
    dg::GaussianZ gaussianZ( 0., p.sigma_z*M_PI, 1); //modulation along fieldline
    if( p.mode == 0 || p.mode == 1)
    {
        dg::Gaussian init0( gp.R_0+p.posX*gp.a, p.posY*gp.a, p.sigma, p.sigma, p.amp);
        if( p.mode == 0)
            y1[1] = asela.ds().fieldaligned().evaluate( init0, gaussianZ, (unsigned)p.Nz/2, 3); //rounds =3 ->2*3-1
        if( p.mode == 1)
            y1[1] = asela.ds().fieldaligned().evaluate( init0, gaussianZ, (unsigned)p.Nz/2, 1); //rounds =1 ->2*1-1
    }
    if( p.mode == 2)
    {
        dg::BathRZ init0(16,16,p.Nz,Rmin,Zmin, 30.,5.,p.amp);
        y1[1] = asela.ds().fieldaligned().evaluate( init0, gaussianZ, (unsigned)p.Nz/2, 1); 
    }
    if( p.mode == 3)
    {
        dg::geo::ZonalFlow<Psip> init0(p.amp, p.k_psi, gp, Psip(gp));
        y1[1] = asela.ds().fieldaligned().evaluate( init0, gaussianZ, (unsigned)p.Nz/2, 1); 
    }
    dg::blas1::axpby( 1., y1[1], 1., y0[1]); //sum up background and perturbation
    dg::blas1::plus(y0[1], -1); //initialize ni-1
    if( p.mode == 2 || p.mode == 3)
    {
        dg::MDVec damping = dg::evaluate( dg::geo::GaussianProfXDamping<Psip>(Psip(gp), gp), grid);
        dg::blas1::pointwiseDot(damping, y0[1], y0[1]); //damp with gaussprofdamp
    }
    std::cout << "intiialize ne" << std::endl;
    if( p.initcond == 0) asela.initializene( y0[1], y0[0]);
    if( p.initcond == 1) dg::blas1::axpby( 1., y0[1], 0.,y0[0], y0[0]); //set n_e = N_i
    std::cout << "Done!\n";    dg::blas1::axpby( 0., y0[2], 0., y0[2]); //set Ue = 0
    dg::blas1::axpby( 0., y0[3], 0., y0[3]); //set Ui = 0
    
    dg::Karniadakis< std::vector<dg::MDVec> > karniadakis( y0, y0[0].size(), p.eps_time);
    karniadakis.init( asela, rolkar, y0, p.dt);
    /////////////////////////////set up netcdf/////////////////////////////////
    file::NC_Error_Handle err;
    int ncid;
    MPI_Info info = MPI_INFO_NULL;
    err = nc_create_par( argv[3], NC_NETCDF4|NC_MPIIO|NC_CLOBBER, comm, info, &ncid); //MPI ON
//     err = nc_create( argv[3],NC_NETCDF4|NC_CLOBBER, &ncid);//MPI OFF

    err = nc_put_att_text( ncid, NC_GLOBAL, "inputfile", input.size(), input.data());
    err = nc_put_att_text( ncid, NC_GLOBAL, "geomfile",  geom.size(), geom.data());
    int dimids[4], tvarID;
    {
        err = file::define_dimensions( ncid, dimids, &tvarID, grid_out.global());

        MagneticField c(gp);
        dg::geo::FieldR<MagneticField> fieldR(c, gp.R_0);
        dg::geo::FieldZ<MagneticField> fieldZ(c, gp.R_0);
        dg::geo::FieldP<MagneticField> fieldP(c, gp.R_0);
        dg::HVec vecR = dg::evaluate( fieldR, grid_out.global());
        dg::HVec vecZ = dg::evaluate( fieldZ, grid_out.global());
        dg::HVec vecP = dg::evaluate( fieldP, grid_out.global());
        int vecID[3];
        err = nc_def_var( ncid, "BR", NC_DOUBLE, 3, &dimids[1], &vecID[0]);
        err = nc_def_var( ncid, "BZ", NC_DOUBLE, 3, &dimids[1], &vecID[1]);
        err = nc_def_var( ncid, "BP", NC_DOUBLE, 3, &dimids[1], &vecID[2]);
        err = nc_enddef( ncid);
        err = nc_put_var_double( ncid, vecID[0], vecR.data());
        err = nc_put_var_double( ncid, vecID[1], vecZ.data());
        err = nc_put_var_double( ncid, vecID[2], vecP.data());
        err = nc_redef(ncid);
    }

    //field IDs 
    std::string names[6] = {"electrons", "ions", "Ue", "Ui", "potential","Aparallel"}; 
    int dataIDs[6]; //VARIABLE IDS
    for( unsigned i=0; i<6; i++)
        err = nc_def_var( ncid, names[i].data(), NC_DOUBLE, 4, dimids, &dataIDs[i]);
    //energy IDs 
    int EtimeID, EtimevarID;
    err = file::define_time( ncid, "energy_time", &EtimeID, &EtimevarID);
    int energyID, massID, energyIDs[6], dissID, alignedID, dEdtID, accuracyID;
    err = nc_def_var( ncid, "energy",   NC_DOUBLE, 1, &EtimeID, &energyID);
    err = nc_def_var( ncid, "mass",   NC_DOUBLE, 1, &EtimeID, &massID);
    std::string energies[6] = {"Se", "Si", "Uperp", "Upare", "Upari","Uapar"}; 
    for( unsigned i=0; i<6; i++)
        err = nc_def_var( ncid, energies[i].data(), NC_DOUBLE, 1, &EtimeID, &energyIDs[i]);
    err = nc_def_var( ncid, "dissipation",   NC_DOUBLE, 1, &EtimeID, &dissID);
    err = nc_def_var( ncid, "alignment",   NC_DOUBLE, 1, &EtimeID, &alignedID);
    err = nc_def_var( ncid, "dEdt",     NC_DOUBLE, 1, &EtimeID, &dEdtID);
    err = nc_def_var( ncid, "accuracy", NC_DOUBLE, 1, &EtimeID, &accuracyID);
    //probe vars definition
    int NepID,phipID;
    err = nc_def_var( ncid, "Ne_p",     NC_DOUBLE, 1, &EtimeID, &NepID);
    err = nc_def_var( ncid, "phi_p",    NC_DOUBLE, 1, &EtimeID, &phipID);  
    for(unsigned i=0; i<6; i++)
    {
        err = nc_var_par_access( ncid, energyIDs[i], NC_COLLECTIVE);
        err = nc_var_par_access( ncid, dataIDs[i], NC_COLLECTIVE);
    }
    err = nc_var_par_access( ncid, tvarID, NC_COLLECTIVE);
    err = nc_var_par_access( ncid, EtimevarID, NC_COLLECTIVE);
    err = nc_var_par_access( ncid, energyID, NC_COLLECTIVE);
    err = nc_var_par_access( ncid, massID, NC_COLLECTIVE);
    err = nc_var_par_access( ncid, dissID, NC_COLLECTIVE);
    err = nc_var_par_access( ncid, alignedID, NC_COLLECTIVE);
    err = nc_var_par_access( ncid, dEdtID, NC_COLLECTIVE);
    err = nc_var_par_access( ncid, accuracyID, NC_COLLECTIVE);
    err = nc_var_par_access( ncid, NepID, NC_COLLECTIVE);
    err = nc_var_par_access( ncid, phipID, NC_COLLECTIVE);
    err = nc_enddef(ncid);
    ///////////////////////////////////PROBE//////////////////////////////
    const dg::HVec Xprobe(1,gp.R_0+p.boxscaleRp*gp.a);
    const dg::HVec Zprobe(1,0.);
    const dg::HVec Phiprobe(1,M_PI);
    dg::IDMatrix probeinterp;
    int probeRANK = grid.pidOf( Xprobe[0], Zprobe[0], Phiprobe[0]);
    if(rank==probeRANK)
        probeinterp=dg::create::interpolation( Xprobe,Zprobe,Phiprobe,grid.local(), dg::NEU);
    dg::DVec probevalue(1,0.);  
    ///////////////////////////first output/////////////////////////////////
    if(rank==0)std::cout << "First output ... \n";
    int dims[3],  coords[3];
    MPI_Cart_get( comm, 3, dims, periods, coords);
    size_t count[4] = {1, grid_out.Nz(), grid_out.n()*(grid_out.Ny()), grid_out.n()*(grid_out.Nx())};
    size_t start[4] = {0, coords[2]*count[1], coords[1]*count[2], coords[0]*count[3]};
    dg::MDVec transfer( dg::evaluate(dg::zero, grid));
    dg::DVec transferD( dg::evaluate(dg::zero, grid_out.local()));
    dg::HVec transferH( dg::evaluate(dg::zero, grid_out.local()));
    dg::IDMatrix interpolate = dg::create::interpolation( grid_out.local(), grid.local()); //create local interpolation matrix
    for( unsigned i=0; i<2; i++)
    {
        dg::blas2::gemv( interpolate, y0[i].data(), transferD);
        dg::blas1::transfer( transferD, transferH);
        err = nc_put_vara_double( ncid, dataIDs[i], start, count, transferH.data() );
    }
    transfer = asela.uparallel()[0];
    dg::blas2::gemv( interpolate, transfer.data(), transferD);
    dg::blas1::transfer( transferD, transferH);
    err = nc_put_vara_double( ncid, dataIDs[2], start, count, transferH.data() );
    transfer = asela.uparallel()[1];
    dg::blas2::gemv( interpolate, transfer.data(), transferD);
    dg::blas1::transfer( transferD, transferH);
    err = nc_put_vara_double( ncid, dataIDs[3], start, count, transferH.data() );
    transfer = asela.potential()[0];
    dg::blas2::gemv( interpolate, transfer.data(), transferD);
    dg::blas1::transfer( transferD, transferH);
    err = nc_put_vara_double( ncid, dataIDs[4], start, count, transferH.data() );
    transfer = asela.aparallel();
    dg::blas2::gemv( interpolate, transfer.data(), transferD);
    dg::blas1::transfer( transferD, transferH);
    err = nc_put_vara_double( ncid, dataIDs[5], start, count, transferH.data() );
    double time = 0;
    err = nc_put_vara_double( ncid, tvarID, start, count, &time);
    err = nc_put_vara_double( ncid, EtimevarID, start, count, &time);

    size_t Estart[] = {0};
    size_t Ecount[] = {1};
    double energy0 = asela.energy(), mass0 = asela.mass(), E0 = energy0, mass = mass0, E1 = 0.0, dEdt = 0., diss = 0., aligned=0, accuracy=0.;
    std::vector<double> evec = asela.energy_vector();
    err = nc_put_vara_double( ncid, energyID, Estart, Ecount, &energy0);
    err = nc_put_vara_double( ncid, massID,   Estart, Ecount, &mass0);
    for( unsigned i=0; i<6; i++)
        err = nc_put_vara_double( ncid, energyIDs[i], Estart, Ecount, &evec[i]);

    err = nc_put_vara_double( ncid, dissID,     Estart, Ecount,&diss);
    err = nc_put_vara_double( ncid, alignedID,  Estart, Ecount,&aligned);
    err = nc_put_vara_double( ncid, dEdtID,     Estart, Ecount,&dEdt);
    err = nc_put_vara_double( ncid, accuracyID, Estart, Ecount,&accuracy);
    //probe
    double Nep=0, phip=0;
    if(rank==probeRANK) {
        dg::blas2::gemv(probeinterp,y0[0].data(),probevalue);
        Nep=probevalue[0] ;
        dg::blas2::gemv(probeinterp,asela.potential()[0].data(),probevalue);
        phip=probevalue[0] ;
    }
    MPI_Bcast( &Nep,1 , MPI_DOUBLE, probeRANK, grid.communicator());
    MPI_Bcast( &phip,1 , MPI_DOUBLE, probeRANK, grid.communicator());
    err = nc_put_vara_double( ncid, NepID,      Estart, Ecount,&Nep);
    err = nc_put_vara_double( ncid, phipID,     Estart, Ecount,&phip);
    if(rank==0)std::cout << "First write successful!\n";
    ///////////////////////////////////////Timeloop/////////////////////////////////
    dg::Timer t;
    t.tic();
#ifdef DG_BENCHMARK
    unsigned step = 0;
#endif //DG_BENCHMARK
    for( unsigned i=1; i<=p.maxout; i++)
    {

#ifdef DG_BENCHMARK
        dg::Timer ti;
        ti.tic();
#endif//DG_BENCHMARK
        for( unsigned j=0; j<p.itstp; j++)
        {
            try{ karniadakis( asela, rolkar, y0);}
            catch( dg::Fail& fail) { 
                if(rank==0)std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
                if(rank==0)std::cerr << "Does Simulation respect CFL condition?"<<std::endl;
                err = nc_close(ncid);
                MPI_Finalize();
                return -1;
            }
            step++;
            time+=p.dt;
            Estart[0] = step;
            E1 = asela.energy(), mass = asela.mass(), diss = asela.energy_diffusion();
            dEdt = (E1 - E0)/p.dt; 
            E0 = E1;
            accuracy = 2.*fabs( (dEdt-diss)/(dEdt + diss));
            evec = asela.energy_vector();
            err = nc_put_vara_double( ncid, EtimevarID, Estart, Ecount, &time);
            err = nc_put_vara_double( ncid, energyID, Estart, Ecount, &E1);
            err = nc_put_vara_double( ncid, massID,   Estart, Ecount, &mass);
            for( unsigned i=0; i<6; i++)
                err = nc_put_vara_double( ncid, energyIDs[i], Estart, Ecount, &evec[i]);
            err = nc_put_vara_double( ncid, dissID,     Estart, Ecount,&diss);
            err = nc_put_vara_double( ncid, alignedID,  Estart, Ecount,&aligned);
            err = nc_put_vara_double( ncid, dEdtID,     Estart, Ecount,&dEdt);
            err = nc_put_vara_double( ncid, accuracyID, Estart, Ecount,&accuracy);
            if(rank==probeRANK)
            {
                dg::blas2::gemv(probeinterp,y0[0].data(),probevalue);
                Nep= probevalue[0] ;
                dg::blas2::gemv(probeinterp,asela.potential()[0].data(),probevalue);
                phip=probevalue[0] ;
            }
            MPI_Bcast( &Nep, 1 ,MPI_DOUBLE, probeRANK, grid.communicator());
            MPI_Bcast( &phip,1 ,MPI_DOUBLE, probeRANK, grid.communicator());
            err = nc_put_vara_double( ncid, NepID,      Estart, Ecount,&Nep);
            err = nc_put_vara_double( ncid, phipID,     Estart, Ecount,&phip);
            if(rank==0)std::cout << "(m_tot-m_0)/m_0: "<< (asela.mass()-mass0)/mass0<<"\t";
            if(rank==0)std::cout << "(E_tot-E_0)/E_0: "<< (E1-energy0)/energy0<<"\t";
            if(rank==0)std::cout <<" d E/dt = " << dEdt <<" Lambda = " << diss << " -> Accuracy: "<< accuracy << "\n";
        }
#ifdef DG_BENCHMARK
        ti.toc();
        if(rank==0)std::cout << "\n\t Step "<<step <<" of "<<p.itstp*p.maxout <<" at time "<<time;
        if(rank==0)std::cout << "\n\t Average time for one step: "<<ti.diff()/(double)p.itstp<<"s";
        ti.tic();
#endif//DG_BENCHMARK
        //err = nc_open_par( argv[3], NC_WRITE|NC_MPIIO, comm, info, &ncid); //dont do it
        //////////////////////////write fields////////////////////////
        start[0] = i;
        for( unsigned j=0; j<2; j++)
        {
            dg::blas2::gemv( interpolate, y0[j].data(), transferD);
            dg::blas1::transfer( transferD, transferH);
            err = nc_put_vara_double( ncid, dataIDs[j], start, count, transferH.data());
        }
    transfer = asela.uparallel()[0];
    dg::blas2::gemv( interpolate, transfer.data(), transferD);
    dg::blas1::transfer( transferD, transferH);
    err = nc_put_vara_double( ncid, dataIDs[2], start, count, transferH.data() );
    transfer = asela.uparallel()[1];
    dg::blas2::gemv( interpolate, transfer.data(), transferD);
    dg::blas1::transfer( transferD, transferH);
    err = nc_put_vara_double( ncid, dataIDs[3], start, count, transferH.data() );
    transfer = asela.potential()[0];
    dg::blas2::gemv( interpolate, transfer.data(), transferD);
    dg::blas1::transfer( transferD, transferH);
    err = nc_put_vara_double( ncid, dataIDs[4], start, count, transferH.data() );
    transfer = asela.aparallel();
    dg::blas2::gemv( interpolate, transfer.data(), transferD);
    dg::blas1::transfer( transferD, transferH);
    err = nc_put_vara_double( ncid, dataIDs[5], start, count, transferH.data() );
        err = nc_put_vara_double( ncid, tvarID, start, count, &time);

        //err = nc_close(ncid); DONT DO IT!
#ifdef DG_BENCHMARK
        ti.toc();
        if(rank==0)std::cout << "\n\t Time for output: "<<ti.diff()<<"s\n\n"<<std::flush;
#endif//DG_BENCHMARK
    }
    t.toc(); 
    unsigned hour = (unsigned)floor(t.diff()/3600);
    unsigned minute = (unsigned)floor( (t.diff() - hour*3600)/60);
    double second = t.diff() - hour*3600 - minute*60;
    if(rank==0)std::cout << std::fixed << std::setprecision(2) <<std::setfill('0');
    if(rank==0)std::cout <<"Computation Time \t"<<hour<<":"<<std::setw(2)<<minute<<":"<<second<<"\n";
    if(rank==0)std::cout <<"which is         \t"<<t.diff()/p.itstp/p.maxout<<"s/step\n";
    err = nc_close(ncid);
    MPI_Finalize();

    return 0;
}
