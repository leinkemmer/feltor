#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <sstream>
#include <cmath>
// #define DG_DEBUG



#include "dg/backend/xspacelib.cuh"
#include "dg/backend/timer.cuh"
#include "dg/backend/interpolation.cuh"
#include "file/nc_utilities.h"
#include "geometries/solovev.h"

#include "asela/asela.cuh"
#include "asela/parameters.h"

/*
   - reads parameters from input.txt or any other given file, 
   - integrates the ToeflR - functor and 
   - writes outputs to a given outputfile using hdf5. 
        density fields are the real densities in XSPACE ( not logarithmic values)
*/

typedef dg::FieldAligned< dg::CylindricalGrid3d, dg::IDMatrix, dg::DVec> DFA;
using namespace dg::geo::solovev;
int main( int argc, char* argv[])
{
    ////////////////////////Parameter initialisation//////////////////////////
    Json::Reader reader;
    Json::Value js, gs;
    if( argc != 4)
    {
        std::cerr << "ERROR: Wrong number of arguments!\nUsage: "<< argv[0]<<" [inputfile] [geomfile] [outputfile]\n";
        return -1;
    }
    else 
    {
        std::ifstream is(argv[1]);
        std::ifstream ks(argv[2]);
        reader.parse(is,js,false);
        reader.parse(ks,gs,false);
    }
    const eule::Parameters p( js);
    const dg::geo::solovev::GeomParameters gp(gs);
    p.display( std::cout);
    gp.display( std::cout);
    std::string input = js.toStyledString(), geom = gs.toStyledString();
    ////////////////////////////////set up computations///////////////////////////

    double Rmin=gp.R_0-p.boxscaleRm*gp.a;
    double Zmin=-p.boxscaleZm*gp.a*gp.elongation;
    double Rmax=gp.R_0+p.boxscaleRp*gp.a; 
    double Zmax=p.boxscaleZp*gp.a*gp.elongation;
    //Make grids
    dg::CylindricalGrid3d grid( Rmin,Rmax, Zmin,Zmax, 0, 2.*M_PI, p.n, p.Nx, p.Ny, 1, p.bc, p.bc, dg::PER);  
    dg::CylindricalGrid3d grid_out( Rmin,Rmax, Zmin,Zmax, 0, 2.*M_PI, p.n_out, p.Nx_out, p.Ny_out,1,p.bc, p.bc, dg::PER);  
    //create RHS 
    std::cout << "Constructing Asela...\n";
    eule::Asela<dg::CylindricalGrid3d, dg::DS<DFA, dg::DMatrix, dg::DVec>, dg::DMatrix, dg::DVec > asela( grid, p, gp); //initialize before rolkar!
    std::cout << "Constructing Rolkar...\n";
    eule::Rolkar<dg::CylindricalGrid3d, dg::DS<DFA, dg::DMatrix, dg::DVec>, dg::DMatrix, dg::DVec> rolkar( grid, p, gp, asela.ds(), asela.dsDIR());
    std::cout << "Done!\n";

    /////////////////////The initial field///////////////////////////////////////////
    //background profile
    dg::geo::Nprofile<Psip> prof(p.bgprofamp, p.nprofileamp, gp, Psip(gp)); //initial background profile
    std::vector<dg::DVec> y0(4, dg::evaluate( prof, grid)), y1(y0); 
    //initial perturbation
    if (p.mode == 0  || p.mode ==1) 
    { 
        dg::Gaussian3d init0( gp.R_0+p.posX*gp.a, p.posY*gp.a, M_PI, p.sigma, p.sigma, p.sigma, p.amp);
        y1[1] = dg::evaluate( init0, grid);
    }
    if (p.mode == 2) 
    { 
        dg::BathRZ init0(16,16,1,Rmin,Zmin, 30.,5.,p.amp);
        y1[1] = dg::evaluate( init0, grid);
    }
    if (p.mode == 3) 
    { 
        dg::geo::ZonalFlow<Psip> init0(p.amp, p.k_psi, gp, Psip(gp));
        y1[1] = dg::evaluate( init0, grid);
    }
    
    dg::blas1::axpby( 1., y1[1], 1., y0[1]); //initialize ni
    dg::blas1::transform(y0[1], y0[1], dg::PLUS<>(-1)); //initialize ni-1
    dg::DVec damping = dg::evaluate( dg::geo::GaussianProfXDamping<Psip>(Psip(gp), gp), grid);
    dg::blas1::pointwiseDot( damping, y0[1], y0[1]); //damp with gaussprofdamp
    std::cout << "initialize ne" << std::endl;
    asela.initializene( y0[1], y0[0]);    
    std::cout << "Done!\n";

    dg::blas1::axpby( 0., y0[2], 0., y0[2]); //set Ue = 0
    dg::blas1::axpby( 0., y0[3], 0., y0[3]); //set Ui = 0
    
    std::cout << "initialize karniadakis" << std::endl;
    dg::Karniadakis< std::vector<dg::DVec> > karniadakis( y0, y0[0].size(), p.eps_time);
    karniadakis.init( asela, rolkar, y0, p.dt);
//     asela.energies(y0); //now energies and potential are at time 0
    std::cout << "Done!\n";
    /////////////////////////////set up netcdf/////////////////////////////////////
    file::NC_Error_Handle err;
    int ncid;
    err = nc_create( argv[3],NC_NETCDF4|NC_CLOBBER, &ncid);
    err = nc_put_att_text( ncid, NC_GLOBAL, "inputfile", input.size(), input.data());
    err = nc_put_att_text( ncid, NC_GLOBAL, "geomfile", geom.size(), geom.data());
    int dim_ids[4], tvarID;
    err = file::define_dimensions( ncid, dim_ids, &tvarID, grid_out);
    MagneticField c(gp);
    dg::geo::FieldR<MagneticField> fieldR(c, gp.R_0);
    dg::geo::FieldZ<MagneticField> fieldZ(c, gp.R_0);
    dg::geo::FieldP<MagneticField> fieldP(c, gp.R_0);
    dg::HVec vecR = dg::evaluate( fieldR, grid_out);
    dg::HVec vecZ = dg::evaluate( fieldZ, grid_out);
    dg::HVec vecP = dg::evaluate( fieldP, grid_out);
    int vecID[3];
    err = nc_def_var( ncid, "BR", NC_DOUBLE, 3, &dim_ids[1], &vecID[0]);
    err = nc_def_var( ncid, "BZ", NC_DOUBLE, 3, &dim_ids[1], &vecID[1]);
    err = nc_def_var( ncid, "BP", NC_DOUBLE, 3, &dim_ids[1], &vecID[2]);
    err = nc_enddef( ncid);
    err = nc_put_var_double( ncid, vecID[0], vecR.data());
    err = nc_put_var_double( ncid, vecID[1], vecZ.data());
    err = nc_put_var_double( ncid, vecID[2], vecP.data());
    err = nc_redef(ncid);

    //field IDs
    std::string names[6] = {"electrons", "ions", "Ue", "Ui", "potential","Aparallel"}; 
    int dataIDs[6]; 
    for( unsigned i=0; i<6; i++){
        err = nc_def_var( ncid, names[i].data(), NC_DOUBLE, 4, dim_ids, &dataIDs[i]);}

    //energy IDs
    int EtimeID, EtimevarID;
    err = file::define_time( ncid, "energy_time", &EtimeID, &EtimevarID);
    int energyID, massID, energyIDs[6], dissID, dEdtID, accuracyID;
    err = nc_def_var( ncid, "energy",   NC_DOUBLE, 1, &EtimeID, &energyID);
    err = nc_def_var( ncid, "mass",   NC_DOUBLE, 1, &EtimeID, &massID);
    std::string energies[6] = {"Se", "Si", "Uperp", "Upare", "Upari","Uapar"}; 
    for( unsigned i=0; i<6; i++){
        err = nc_def_var( ncid, energies[i].data(), NC_DOUBLE, 1, &EtimeID, &energyIDs[i]);}
    err = nc_def_var( ncid, "dissipation",   NC_DOUBLE, 1, &EtimeID, &dissID);
    err = nc_def_var( ncid, "dEdt",     NC_DOUBLE, 1, &EtimeID, &dEdtID);
    err = nc_def_var( ncid, "accuracy", NC_DOUBLE, 1, &EtimeID, &accuracyID);
    err = nc_enddef(ncid);
    ///////////////////////////////////first output/////////////////////////
    std::cout << "First output ... \n";
    size_t start[4] = {0, 0, 0, 0};
    size_t count[4] = {1, grid_out.Nz(), grid_out.n()*grid_out.Ny(), grid_out.n()*grid_out.Nx()};
    dg::DVec transfer(  dg::evaluate(dg::zero, grid));
    dg::DVec transferD( dg::evaluate(dg::zero, grid_out));
    dg::HVec transferH( dg::evaluate(dg::zero, grid_out));
    dg::IDMatrix interpolate = dg::create::interpolation( grid_out, grid); 
    for( unsigned i=0; i<2; i++)
    {
        dg::blas2::symv( interpolate, y0[i], transferD);
        dg::blas1::transfer( transferD, transferH);
        err = nc_put_vara_double( ncid, dataIDs[i], start, count, transferH.data() );
    }
    transfer = asela.uparallel()[0];
    dg::blas2::symv( interpolate, transfer, transferD);
    dg::blas1::transfer( transferD, transferH);
    err = nc_put_vara_double( ncid, dataIDs[2], start, count, transferH.data() );
    transfer = asela.uparallel()[1];
    dg::blas2::symv( interpolate, transfer, transferD);
    dg::blas1::transfer( transferD, transferH);
    err = nc_put_vara_double( ncid, dataIDs[3], start, count, transferH.data() );
    transfer = asela.potential()[0];
    dg::blas2::symv( interpolate, transfer, transferD);
    dg::blas1::transfer( transferD, transferH);
    err = nc_put_vara_double( ncid, dataIDs[4], start, count, transferH.data() );
    transfer = asela.aparallel();
    dg::blas2::symv( interpolate, transfer, transferD);
    dg::blas1::transfer( transferD, transferH);
    err = nc_put_vara_double( ncid, dataIDs[5], start, count, transferH.data() );
    double time = 0;
    err = nc_put_vara_double( ncid, tvarID, start, count, &time);
    err = nc_put_vara_double( ncid, EtimevarID, start, count, &time);

    size_t Estart[] = {0};
    size_t Ecount[] = {1};
    double energy0 = asela.energy(), mass0 = asela.mass(), E0 = energy0, mass = mass0, E1 = 0.0, dEdt = 0., diss = 0., accuracy=0.;
    std::vector<double> evec = asela.energy_vector();
    err = nc_put_vara_double( ncid, energyID, Estart, Ecount, &energy0);
    err = nc_put_vara_double( ncid, massID,   Estart, Ecount, &mass0);
    for( unsigned i=0; i<6; i++)
        err = nc_put_vara_double( ncid, energyIDs[i], Estart, Ecount, &evec[i]);

    err = nc_put_vara_double( ncid, dissID,     Estart, Ecount,&diss);
    err = nc_put_vara_double( ncid, dEdtID,     Estart, Ecount,&dEdt);
    err = nc_put_vara_double( ncid, accuracyID, Estart, Ecount,&accuracy);
    err = nc_close(ncid);
    std::cout << "First write successful!\n";

    ///////////////////////////////////////Timeloop/////////////////////////////////
    dg::Timer t;
    t.tic();
#ifdef DG_BENCHMARK
    unsigned step = 0;
#endif //DG_BENCHMARK
    for( unsigned i=1; i<=p.maxout; i++)
    {

#ifdef DG_BENCHMARK
        dg::Timer ti;
        ti.tic();
#endif//DG_BENCHMARK
        for( unsigned j=0; j<p.itstp; j++)
        {
            try{ karniadakis( asela, rolkar, y0);}
            catch( dg::Fail& fail) { 
                std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
                std::cerr << "Does Simulation respect CFL condition?\n";
                err = nc_close(ncid);
                return -1;
            }
            step++;
            time+=p.dt;
//             asela.energies(y0);//advance potential and energies
            Estart[0] = step;
            E1 = asela.energy(), mass = asela.mass(), diss = asela.energy_diffusion();
            dEdt = (E1 - E0)/p.dt; 
            E0 = E1;
            accuracy = 2.*fabs( (dEdt-diss)/(dEdt + diss));
            evec = asela.energy_vector();
            err = nc_open(argv[3], NC_WRITE, &ncid);
            err = nc_put_vara_double( ncid, EtimevarID, Estart, Ecount, &time);
            err = nc_put_vara_double( ncid, energyID, Estart, Ecount, &E1);
            err = nc_put_vara_double( ncid, massID,   Estart, Ecount, &mass);
            for( unsigned i=0; i<6; i++)
            {
                err = nc_put_vara_double( ncid, energyIDs[i], Estart, Ecount, &evec[i]);
            }
            err = nc_put_vara_double( ncid, dissID,     Estart, Ecount,&diss);
            err = nc_put_vara_double( ncid, dEdtID,     Estart, Ecount,&dEdt);
            err = nc_put_vara_double( ncid, accuracyID, Estart, Ecount,&accuracy);
            std::cout << "(m_tot-m_0)/m_0: "<< (asela.mass()-mass0)/mass0<<"\t";
            std::cout << "(E_tot-E_0)/E_0: "<< (E1-energy0)/energy0<<"\t";
            std::cout <<" d E/dt = " << dEdt <<" Lambda = " << diss << " -> Accuracy: "<< accuracy << "\n";
            err = nc_close(ncid);

        }
#ifdef DG_BENCHMARK
        ti.toc();
        std::cout << "\n\t Step "<<step <<" of "<<p.itstp*p.maxout <<" at time "<<time;
        std::cout << "\n\t Average time for one step: "<<ti.diff()/(double)p.itstp<<"s\n\n"<<std::flush;
#endif//DG_BENCHMARK
        //////////////////////////write fields////////////////////////
        start[0] = i;
        err = nc_open(argv[3], NC_WRITE, &ncid);
        for( unsigned j=0; j<2; j++)
        {
            dg::blas2::symv( interpolate, y0[j], transferD);
            dg::blas1::transfer( transferD, transferH);
            err = nc_put_vara_double( ncid, dataIDs[j], start, count, transferH.data());
        }
        transfer = asela.uparallel()[0];
        dg::blas2::symv( interpolate, transfer, transferD);
        dg::blas1::transfer( transferD, transferH);
        err = nc_put_vara_double( ncid, dataIDs[2], start, count, transferH.data() );
        transfer = asela.uparallel()[1];
        dg::blas2::symv( interpolate, transfer, transferD);
        dg::blas1::transfer( transferD, transferH);
        err = nc_put_vara_double( ncid, dataIDs[3], start, count, transferH.data() );
        transfer = asela.potential()[0];
        dg::blas2::symv( interpolate, transfer, transferD);
        dg::blas1::transfer( transferD, transferH);
        err = nc_put_vara_double( ncid, dataIDs[4], start, count, transferH.data() );
        transfer = asela.aparallel();
        dg::blas2::symv( interpolate, transfer, transferD);
        dg::blas1::transfer( transferD, transferH);
        err = nc_put_vara_double( ncid, dataIDs[5], start, count, transferH.data() );
        err = nc_put_vara_double( ncid, tvarID, start, count, &time);
        err = nc_close(ncid);
    }
    t.toc(); 
    unsigned hour = (unsigned)floor(t.diff()/3600);
    unsigned minute = (unsigned)floor( (t.diff() - hour*3600)/60);
    double second = t.diff() - hour*3600 - minute*60;
    std::cout << std::fixed << std::setprecision(2) <<std::setfill('0');
    std::cout <<"Computation Time \t"<<hour<<":"<<std::setw(2)<<minute<<":"<<second<<"\n";
    std::cout <<"which is         \t"<<t.diff()/p.itstp/p.maxout<<"s/step\n";

    return 0;

}

