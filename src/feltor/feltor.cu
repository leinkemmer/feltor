#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <sstream>
#include <cmath>
// #define DG_DEBUG

#include "draw/host_window.h"
//#include "draw/device_window.cuh"
#include "dg/backend/xspacelib.cuh"
#include "dg/backend/sparseblockmat.cuh"
#include "dg/backend/timer.cuh"
#include "dg/backend/average.cuh"
#include "dg/backend/typedefs.cuh"

#include "feltor.cuh"

/*
   - reads parameters from input.txt or any other given file, 
   - integrates the Explicit - functor and 
   - directly visualizes results on the screen using parameters in window_params.txt
*/

int main( int argc, char* argv[])
{
    ////Parameter initialisation ////////////////////////////////////////////
    Json::Reader reader;
    Json::Value js, gs;
    if( argc == 1)
    {
        std::ifstream is("input.json");
        std::ifstream ks("geometry_params.json");
        reader.parse(is,js,false);
        reader.parse(ks,gs,false);
    }
    else if( argc == 3)
    {
        std::ifstream is(argv[1]);
        std::ifstream ks(argv[2]);
        reader.parse(is,js,false);
        reader.parse(ks,gs,false);
    }
    else
    {
        std::cerr << "ERROR: Too many arguments!\nUsage: "<< argv[0]<<" [inputfile] [geomfile] \n";
        return -1;
    }
    const feltor::Parameters p( js);
    const dg::geo::solovev::GeomParameters gp(gs);
    p.display( std::cout);
    gp.display( std::cout);
    /////////glfw initialisation ////////////////////////////////////////////
    std::stringstream title;
    std::ifstream is( "window_params.js");
    reader.parse( is, js, false);
    is.close();
    unsigned red = js.get("reduction", 1).asUInt();
    GLFWwindow* w = draw::glfwInitAndCreateWindow( (p.Nz/red+1)*js["width"].asDouble(), js["rows"].asDouble()*js["height"].asDouble(), "");
    draw::RenderHostData render(js["rows"].asDouble(), p.Nz/red + 1);
    /////////////////////////////////////////////////////////////////////////
    double Rmin=gp.R_0-p.boxscaleRm*gp.a;
    double Zmin=-p.boxscaleZm*gp.a*gp.elongation;
    double Rmax=gp.R_0+p.boxscaleRp*gp.a; 
    double Zmax=p.boxscaleZp*gp.a*gp.elongation;
    //Make grid
    dg::CylindricalGrid3d grid( Rmin,Rmax, Zmin,Zmax, 0, 2.*M_PI, p.n, p.Nx, p.Ny, p.Nz, p.bc, p.bc, dg::PER);  

    //create RHS 
    std::cout << "Constructing Explicit...\n";
    feltor::Explicit<dg::CylindricalGrid3d, dg::IDMatrix, dg::DMatrix, dg::DVec> feltor( grid, mag, p, gp); //initialize before rolkar!
    std::cout << "Constructing Implicit...\n";
    feltor::Implicit<dg::CylindricalGrid3d, dg::IDMatrix, dg::DMatrix, dg::DVec> rolkar( grid, p, gp, feltor.ds(), feltor.dsDIR());
    std::cout << "Done!\n";

    /////////////////////The initial field///////////////////////////////////////////
    //background profile
    dg::geo::Nprofile<Psip> prof(p.bgprofamp, p.nprofileamp, gp, Psip(gp)); //initial background profile
    std::vector<dg::DVec> y0(4, dg::evaluate( prof, grid)), y1(y0); 
    //perturbation 
    dg::GaussianZ gaussianZ( 0., p.sigma_z*M_PI, 1); //modulation along fieldline
    if( p.mode == 0 || p.mode == 1)
    {
        dg::Gaussian init0( gp.R_0+p.posX*gp.a, p.posY*gp.a, p.sigma, p.sigma, p.amp);
        if( p.mode == 0)
            y1[1] = feltor.ds().fieldaligned().evaluate( init0, gaussianZ, (unsigned)p.Nz/2, 3); //rounds =3 ->2*3-1
        if( p.mode == 1)
            y1[1] = feltor.ds().fieldaligned().evaluate( init0, gaussianZ, (unsigned)p.Nz/2, 1); //rounds =1 ->2*1-1
    }
    if( p.mode == 2)
    {
        dg::BathRZ init0(16,16,p.Nz,Rmin,Zmin, 30.,5.,p.amp);
        y1[1] = feltor.ds().fieldaligned().evaluate( init0, gaussianZ, (unsigned)p.Nz/2, 1); 
    }
    if( p.mode == 3)
    {
        dg::geo::ZonalFlow<Psip> init0(p.amp, p.k_psi, gp, Psip(gp));
        y1[1] = feltor.ds().fieldaligned().evaluate( init0, gaussianZ, (unsigned)p.Nz/2, 1); 
    }
    dg::blas1::axpby( 1., y1[1], 1., y0[1]); //sum up background and perturbation
    dg::blas1::plus(y0[1], -1); //initialize ni-1
    if( p.mode == 2 || p.mode == 3)
    {
        dg::DVec damping = dg::evaluate( dg::geo::GaussianProfXDamping<Psip>(Psip(gp), gp), grid);
        dg::blas1::pointwiseDot(damping,y0[1], y0[1]); //damp with gaussprofdamp
    }
    std::cout << "intiialize ne" << std::endl;
    if( p.initcond == 0) feltor.initializene( y0[1], y0[0]);
    if( p.initcond == 1) dg::blas1::axpby( 1., y0[1], 0.,y0[0], y0[0]); //set n_e = N_i
    std::cout << "Done!\n";

    dg::blas1::axpby( 0., y0[2], 0., y0[2]); //set Ue = 0
    dg::blas1::axpby( 0., y0[3], 0., y0[3]); //set Ui = 0

    dg::Karniadakis< std::vector<dg::DVec> > karniadakis( y0, y0[0].size(), p.eps_time);
    std::cout << "intiialize karniadakis" << std::endl;
    karniadakis.init( feltor, rolkar, y0, p.dt);
    std::cout << "Done!\n";
//     feltor.energies( y0);//now energies and potential are at time 0

    dg::DVec dvisual( grid.size(), 0.);
    dg::HVec hvisual( grid.size(), 0.), visual(hvisual),avisual(hvisual);
    dg::IHMatrix equi = dg::create::backscatter( grid);
    draw::ColorMapRedBlueExtMinMax colors(-1.0, 1.0);
    dg::ToroidalAverage<dg::HVec> toravg(grid);
    //create timer
    dg::Timer t;
    double time = 0;
    unsigned step = 0;
    
    const double mass0 = feltor.mass();
    double E0 = feltor.energy(), energy0 = E0, E1 = 0., dEdt = 0.;
    
    std::cout << "Begin computation \n";
    std::cout << std::scientific << std::setprecision( 2);
     //probe
    const dg::HVec Xprobe(1,gp.R_0+p.boxscaleRp*gp.a);
    const dg::HVec Zprobe(1,0.);
    const dg::HVec Phiprobe(1,M_PI);
    dg::IDMatrix probeinterp;
    dg::blas2::transfer( dg::create::interpolation( Xprobe, Zprobe, Phiprobe, grid, dg::NEU), probeinterp);
    dg::DVec probevalue(1,0.);   
    while ( !glfwWindowShouldClose( w ))
    {

        dg::blas1::transfer( y0[0], hvisual);
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (double)thrust::reduce( visual.begin(), visual.end(), 0., thrust::maximum<double>() );
        colors.scalemin() = -colors.scalemax();        
        //colors.scalemin() = 1.0;
        //colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() );

        title << std::setprecision(2) << std::scientific;
        //title <<"ne / "<<(double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() )<<"  " << colors.scalemax()<<"\t";
        title <<"ne-1 / " << colors.scalemax()<<"\t";
        for( unsigned k=0; k<p.Nz/red;k++)
        {
            unsigned size=grid.n()*grid.n()*grid.Nx()*grid.Ny();
            dg::HVec part( visual.begin() + k*red*size, visual.begin()+(k*red+1)*size);   
            render.renderQuad( part, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        }
        dg::blas1::axpby(0.0,avisual,0.0,avisual);
        toravg(visual,avisual);
        render.renderQuad( avisual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        //draw ions
        //thrust::transform( y1[1].begin(), y1[1].end(), dvisual.begin(), dg::PLUS<double>(-0.));//ne-1
        dg::blas1::transfer( y0[1], hvisual);
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (double)thrust::reduce( visual.begin(), visual.end(), 0., thrust::maximum<double>() );
        //colors.scalemin() = 1.0;        
        colors.scalemin() = -colors.scalemax();        
        //colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() );

        title << std::setprecision(2) << std::scientific;
        //title <<"ni / "<<(double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() )<<"  " << colors.scalemax()<<"\t";
        title <<"ni-1 / " << colors.scalemax()<<"\t";
        for( unsigned k=0; k<p.Nz/red;k++)
        {
            unsigned size=grid.n()*grid.n()*grid.Nx()*grid.Ny();
            dg::HVec part( visual.begin() + k*red*size, visual.begin()+(k*red+1)*size);
            render.renderQuad( part, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        }
        dg::blas1::axpby(0.0,avisual,0.0,avisual);
        toravg(visual,avisual);
        render.renderQuad( avisual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        
        //draw potential
        //transform to Vor
        dvisual=feltor.potential()[0];
        dg::blas2::gemv( rolkar.laplacianM(), dvisual, y1[1]);
        dg::blas1::transfer( y1[1], hvisual);
//         hvisual = feltor.potential()[0];
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (double)thrust::reduce( visual.begin(),visual.end(), 0.,thrust::maximum<double>()  );
//         colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() );
        colors.scalemin() = -colors.scalemax();
        //title <<"Phi / "<<colors.scalemin()<<"  " << colors.scalemax()<<"\t";
        title <<"Omega / "<< colors.scalemax()<<"\t";
        for( unsigned k=0; k<p.Nz/red;k++)
        {
            unsigned size=grid.n()*grid.n()*grid.Nx()*grid.Ny();
            dg::HVec part( visual.begin() + k*red*size, visual.begin()+(k*red+1)*size);
            render.renderQuad( part, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        }
        dg::blas1::axpby(0.0,avisual,0.0,avisual);
        toravg(visual,avisual);
        render.renderQuad( avisual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        
        //draw U_e
        dg::blas1::transfer( y0[2], hvisual);
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (double)thrust::reduce( visual.begin(), visual.end(), 0.,thrust::maximum<double>()  );
        //colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() );
        colors.scalemin() = -colors.scalemax();
        //title <<"Ue / "<<colors.scalemin()<<"  " << colors.scalemax()<<"\t";
        title <<"Ue / " << colors.scalemax()<<"\t";
                for( unsigned k=0; k<p.Nz/red;k++)
        {
            unsigned size=grid.n()*grid.n()*grid.Nx()*grid.Ny();
            dg::HVec part( visual.begin() + k*red*size, visual.begin()+(k*red+1)*size);
            render.renderQuad( part, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        }
        dg::blas1::axpby(0.0,avisual,0.0,avisual);
        toravg(visual,avisual);
        render.renderQuad( avisual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);      
        
        //draw U_i
        dg::blas1::transfer( y0[3], hvisual);
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (double)thrust::reduce( visual.begin(), visual.end(), 0., thrust::maximum<double>()  );
        //colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() );
        colors.scalemin() = -colors.scalemax();
        //title <<"Ui / "<<colors.scalemin()<< "  " << colors.scalemax()<<"\t";
        title <<"Ui / " << colors.scalemax()<<"\t";
        for( unsigned k=0; k<p.Nz/red;k++)
        {
            unsigned size=grid.n()*grid.n()*grid.Nx()*grid.Ny();
            dg::HVec part( visual.begin() + k*red*size, visual.begin()+(k*red+1)*size);
            render.renderQuad( part, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        }
        dg::blas1::axpby(0.0,avisual,0.0,avisual);
        toravg(visual,avisual);
        render.renderQuad( avisual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        
        title << std::fixed; 
        title << " &&   time = "<<time;
        glfwSetWindowTitle(w,title.str().c_str());
        title.str("");
        glfwPollEvents();
        glfwSwapBuffers( w);

        //step 
#ifdef DG_BENCHMARK
        t.tic();
#endif//DG_BENCHMARK
        for( unsigned i=0; i<p.itstp; i++)
        {
            try{ karniadakis( feltor, rolkar, y0);}
            catch( dg::Fail& fail) { 
                std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
                std::cerr << "Does Simulation respect CFL condition?\n";
                glfwSetWindowShouldClose( w, GL_TRUE);
                break;
            }
            step++;
//             feltor.energies( y0); //update energetics
            //Compute probe values
            dg::blas2::gemv(probeinterp,y0[0],probevalue);
            std::cout << " Ne_p - 1  = " << probevalue[0] <<"\t";
            dg::blas2::gemv(probeinterp,feltor.potential()[0],probevalue);
            std::cout << " Phi_p = " << probevalue[0] <<"\t";
            std::cout << "(m_tot-m_0)/m_0: "<< (feltor.mass()-mass0)/mass0<<"\t";
            E1 = feltor.energy();
            dEdt = (E1 - E0)/p.dt; //
            double diss = feltor.energy_diffusion( );
            std::cout << "(E_tot-E_0)/E_0: "<< (E1-energy0)/energy0<<"\t";
            std::cout << "Accuracy: "<< 2.*fabs((dEdt-diss)/(dEdt+diss))<<" d E/dt = " << dEdt <<" Lambda =" << diss << "\n";
            
            E0 = E1;

        }
        time += (double)p.itstp*p.dt;
#ifdef DG_BENCHMARK
        t.toc();
        std::cout << "\n\t Step "<<step;
        std::cout << "\n\t Average time for one step: "<<t.diff()/(double)p.itstp<<"s\n\n";
#endif//DG_BENCHMARK
    }
    glfwTerminate();
    ////////////////////////////////////////////////////////////////////

    return 0;

}
