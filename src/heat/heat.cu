#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <sstream>
#include <cmath>
// #define DG_DEBUG

#include "draw/host_window.h"
//#include "draw/device_window.cuh"
#include "dg/backend/xspacelib.cuh"
#include "dg/backend/timer.cuh"
#include "dg/runge_kutta.h"
#include "dg/multistep.h"
#include "dg/elliptic.h"
#include "dg/cg.h"

#include "geometries/geometries.h"
#include "heat/parameters.h"


#include "heat.cuh"

typedef dg::FieldAligned< dg::CylindricalGrid3d, dg::IDMatrix, dg::DVec> DFA;
using namespace dg::geo::solovev;

int main( int argc, char* argv[])
{
    ////Parameter initialisation ////////////////////////////////////////////
    std::stringstream title;
    Json::Reader reader;
    Json::Value js, gs;
    if( argc == 1)
    {
        std::ifstream is("input.json");
        std::ifstream ks("geometry_params.js");
        reader.parse(is,js,false);
        reader.parse(ks,gs,false);
    }
    else if( argc == 3)
    {
        std::ifstream is(argv[1]);
        std::ifstream ks(argv[2]);
        reader.parse(is,js,false);
        reader.parse(ks,gs,false);
    }
    else
    {
        std::cerr << "ERROR: Wrong number of arguments!\nUsage: "<< argv[0]<<" [inputfile] [geomfile] \n";
        return -1;
    }
    const eule::Parameters p( js); p.display( std::cout);
    const GeomParameters gp(gs); gp.display( std::cout);
    /////////glfw initialisation ////////////////////////////////////////////
    std::ifstream is( "window_params.js");
    reader.parse( is, js, false);
    is.close();
    GLFWwindow* w = draw::glfwInitAndCreateWindow( js["width"].asDouble(), js["height"].asDouble(), "");
    draw::RenderHostData render(js["rows"].asDouble(), js["cols"].asDouble());
    /////////////////////////////////////////////////////////////////////////
    
    double Rmin=gp.R_0-p.boxscaleRm*gp.a;
    double Zmin=-p.boxscaleZm*gp.a*gp.elongation;
    double Rmax=gp.R_0+p.boxscaleRp*gp.a; 
    double Zmax=p.boxscaleZp*gp.a*gp.elongation;

     dg::CylindricalGrid3d grid( Rmin,Rmax, Zmin,Zmax, 0, 2.*M_PI, p.n, p.Nx, p.Ny, p.Nz, p.bc, p.bc, dg::PER);  

//     dg::DVec w3d_ = dg::create::volume( grid);
//     dg::DVec v3d_ = dg::create::inv_volume( grid);
//     dg::DVec x = dg::evaluate( dg::zero, grid);
//     set up the parallel diffusion

    
//     dg::GeneralEllipticSym<dg::DMatrix, dg::DVec, dg::DVec> ellipticsym( grid, dg::not_normed, dg::forward);
//     dg::DVec bfield = dg::evaluate( solovev::bR( gp.R_0, gp.I_0),grid);
// 
//     ellipticsym.set_x( bfield);
//     bfield = dg::evaluate( solovev::bZ( gp.R_0, gp.I_0),grid);
//     ellipticsym.set_y( bfield);
//     bfield = dg::evaluate( solovev::bPhi( gp.R_0, gp.I_0),grid);
//     ellipticsym.set_z( bfield);
// //     
//     
//     double eps =1e-12;   
//     dg::Invert< dg::DVec> invert( x, w3d_.size(), eps );  
//     std::cout << "MAX # iterations = " << w3d_.size() << std::endl;
//     const dg::DVec rhs = dg::evaluate( solovev::DeriNeuT2( gp.R_0, gp.I_0), grid);
//     std::cout << " # of iterations "<< invert( ellipticsym, x, rhs ) << std::endl; //is dsTds
//     dg::DVec solution = dg::evaluate( solovev::FuncNeu(gp.R_0, gp.I_0),grid);
//     double normf = dg::blas2::dot( w3d_, solution);
//     std::cout << "Norm analytic Solution  "<<sqrt( normf)<<"\n";
//     double errinvT =dg::blas2::dot( w3d_, x);
//     std::cout << "Norm numerical Solution "<<sqrt( errinvT)<<"\n";
//     dg::blas1::axpby( 1., solution, +1.,x);
//     errinvT =dg::blas2::dot( w3d_, x);
//     std::cout << "Relative Difference is  "<< sqrt( errinvT/normf )<<"\n";

    
        
//     std::cout << "MAX # iterations = " << w3d_.size() << std::endl;
//     
//     std::cout << " # of iterations "<< invert( ellipticsym, x, rhs ) << std::endl; //is dsTds
//     
//     std::cout << "Norm analytic Solution  "<<sqrt( normf)<<"\n";
//     errinvT =dg::blas2::dot( w3d_, x);
//     std::cout << "Norm numerical Solution "<<sqrt( errinvT)<<"\n";
//     dg::blas1::axpby( 1., solution, +1.,x);
//     errinvT =dg::blas2::dot( w3d_, x);
//     std::cout << "Relative Difference is  "<< sqrt( errinvT/normf )<<"\n";
// 
   

    
    //create RHS     
    std::cout << "initialize feltor" << std::endl;
    eule::Feltor<dg::DS<DFA, dg::DMatrix, dg::DVec>, dg::DMatrix, dg::DVec > feltor( grid, p,gp); //initialize before rolkar!
    std::cout << "initialize rolkar" << std::endl;
    eule::Rolkar<dg::CylindricalGrid3d , dg::DS<DFA, dg::DMatrix, dg::DVec>, dg::DMatrix, dg::DVec > rolkar( grid, p,gp);

    ////////////////////////////////The initial field////////////////////////////////
 //initial perturbation
//     std::cout << "initialize delta T" << std::endl;
    dg::Gaussian3d init0(gp.R_0+p.posX*gp.a, p.posY*gp.a, M_PI, p.sigma, p.sigma, p.sigma_z, p.amp);
//     dg::Gaussian init0( gp.R_0+p.posX*gp.a, p.posY*gp.a, p.sigma, p.sigma, p.amp);
//     dg::BathRZ init0(16,16,p.Nz,Rmin,Zmin, 30.,5.,p.amp);
//     solovev::ZonalFlow init0(p, gp);

//     dg::CONSTANT init0( 0.);

    
    //background profile
    std::cout << "T background" << std::endl;
    dg::geo::Nprofile<Psip> prof(p.bgprofamp, p.nprofileamp, gp, Psip(gp)); //initial background profile
    std::vector<dg::DVec> y0(1, dg::evaluate( prof, grid)), y1(y0); 
    
//     //field aligning
    std::cout << "T aligning" << std::endl;  
//     dg::CONSTANT gaussianZ( 1.);
    dg::GaussianZ gaussianZ( 0., p.sigma_z*M_PI, 1);
    y1[0] = feltor.ds().fieldaligned().evaluate( init0, gaussianZ, (unsigned)p.Nz/2, 3); //rounds =2 ->2*2-1 //3 rounds for blob

    //no field aligning
//     std::cout << "No T aligning" << std::endl;      
//     y1[0] = dg::evaluate( init0, grid);
//        dg::blas1::pointwiseDot(rolkar.damping(),y1[0], y1[0]); //damp with gaussprofdamp
 
    dg::blas1::axpby( 1., y1[0], 1., y0[0]); //initialize ni
    if (p.bc ==dg::DIR)    {
    dg::blas1::transform(y0[0], y0[0], dg::PLUS<>(-1)); //initialize ni-1
    }

//     dg::blas1::pointwiseDot(rolkar.damping(),y0[0], y0[0]); //damp with gaussprofdamp
    std::cout << "Done!\n";

    //////////////////////////////////////////////////////////////////////////////////
    //RK solver
//     dg::RK<4, std::vector<dg::DVec> >  rk( y0);
    //SIRK solver
    dg::SIRK<std::vector<dg::DVec> > sirk(y0, grid.size(),p.eps_time);
//     dg::Karniadakis< std::vector<dg::DVec> > karniadakis( y0, y0[0].size(),1e-13);
//     karniadakis.init( feltor, rolkar, y0, p.dt);

     feltor.energies( y0);//now energies and potential are at time 0
   
    dg::DVec dvisual( grid.size(), 0.);
    dg::HVec hvisual( grid.size(), 0.), visual(hvisual),avisual(hvisual);
    dg::IHMatrix equi = dg::create::backscatter( grid);
    draw::ColorMapRedBlueExtMinMax colors(-1.0, 1.0);

    //create timer
    dg::Timer t;
    double time = 0;
    unsigned step = 0;
    
    const double mass0 = feltor.mass(), mass_blob0 = mass0 - grid.lx()*grid.ly();
    double E0 = feltor.energy(), energy0 = E0, E1 = 0, diff = 0;
    std::cout << "Begin computation \n";
    std::cout << std::scientific << std::setprecision( 2);
    dg::DVec T0 = dg::evaluate( dg::one, grid);  
    dg::DVec T1 = dg::evaluate( dg::one, grid);  

    dg::blas1::axpby( 1., y0[0], 0., T0); //initialize ni
    dg::DVec w3d =  dg::create::volume(grid);
    double normT0 = dg::blas2::dot(  w3d, T0);
    while ( !glfwWindowShouldClose( w ))
    {
        dg::blas1::transfer( y0[0], hvisual);
        if (p.bc ==dg::NEU)    {
        dg::blas1::transform(hvisual,hvisual , dg::PLUS<>(-1)); //npe = N+1
        }
        dg::blas2::gemv( equi, hvisual, visual);
        dg::blas1::axpby(0.0,avisual,0.0,avisual);
        for( unsigned k=0; k<p.Nz;k++)
        {
            unsigned size=grid.n()*grid.n()*grid.Nx()*grid.Ny();
            dg::HVec part( visual.begin() + k*size, visual.begin()+(k+1)*size);
            dg::blas1::axpby(1.0,part,1.0,avisual);
//             render.renderQuad( part, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        }
        dg::blas1::scal(avisual,1./p.Nz);
        colors.scalemax() = (double)thrust::reduce( avisual.begin(), avisual.end(), 0., thrust::maximum<double>() );
        colors.scalemin() = -colors.scalemax();        
//                 colors.scalemin() =  (double)thrust::reduce( avisual.begin(), avisual.end(), colors.scalemax()  ,thrust::minimum<double>() );
        title << std::setprecision(2) << std::scientific;
        title <<"T-1 / " << colors.scalemax()<<"\t";
        render.renderQuad( avisual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);   
        
        title << std::fixed; 
        title << " &&   time = "<<time;
        glfwSetWindowTitle(w,title.str().c_str());
        title.str("");
        glfwPollEvents();
        glfwSwapBuffers( w);

        //step 
#ifdef DG_BENCHMARK
        t.tic();
#endif//DG_BENCHMARK
        for( unsigned i=0; i<p.itstp; i++)
        {
            step++;
            feltor.energies( y0); //update energetics
            std::cout << "(m_tot-m_0)/m_0: "<< (feltor.mass()-mass0)/mass_blob0<<"\t";
            E1 = feltor.energy();
            diff = (E1 - E0)/p.dt; //
            double diss = feltor.energy_diffusion( );
            dg::blas1::axpby( 1., y0[0], -1.,T0, T1);
            double err = sqrt(dg::blas2::dot( w3d, T1)/normT0);
            std::cout << "(E_tot-E_0)/E_0: "<< (E1-energy0)/energy0<<"\t";
            std::cout << "Accuracy: "<< 2.*(diff-diss)/(diff+diss)<<" d E/dt = " << diff <<" Lambda =" << diss << " err =" << err << "\n";
            E0 = E1;
            try{
//                 rk( feltor, y0, y1, p.dt);
                 sirk(feltor,rolkar,y0,y1,p.dt);
//                 karniadakis( feltor, rolkar, y0);

                y0.swap( y1);}
              catch( dg::Fail& fail) { 
                std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
                std::cerr << "Does Simulation respect CFL condition?\n";
                glfwSetWindowShouldClose( w, GL_TRUE);
                break;}


        }
        time += (double)p.itstp*p.dt;
#ifdef DG_BENCHMARK
        t.toc();
        std::cout << "\n\t Step "<<step;
        std::cout << "\n\t Average time for one step: "<<t.diff()/(double)p.itstp<<"s\n\n";
#endif//DG_BENCHMARK
    }
    
    glfwTerminate();


    ////////////////////////////////////////////////////////////////////

    return 0;

}
