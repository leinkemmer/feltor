#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <sstream>

#include "draw/host_window.h"
//#include "draw/device_window.cuh"

#include "toeflR.cuh"
#include "dg/algorithm.h"
#include "dg/backend/xspacelib.cuh"
#include "parameters.h"

/*
   - reads parameters from input.txt or any other given file, 
   - integrates the ToeflR - functor and 
   - directly visualizes results on the screen using parameters in window_params.txt
*/


int main( int argc, char* argv[])
{
    ////Parameter initialisation ////////////////////////////////////////////
    std::stringstream title;
    Json::Reader reader;
    Json::Value js;
    if( argc == 1)
    {
        std::ifstream is("input.json");
        reader.parse(is,js,false);
    }
    else if( argc == 2)
    {
        std::ifstream is(argv[1]);
        reader.parse(is,js,false);
    }
    else
    {
        std::cerr << "ERROR: Too many arguments!\nUsage: "<< argv[0]<<" [filename]\n";
        return -1;
    }
    const Parameters p( js);
    p.display( std::cout);
    /////////glfw initialisation ////////////////////////////////////////////
    std::ifstream is( "window_params.js");
    reader.parse( is, js, false);
    is.close();
    GLFWwindow* w = draw::glfwInitAndCreateWindow( js["width"].asDouble(), js["height"].asDouble(), "");
    draw::RenderHostData render(js["rows"].asDouble(), js["cols"].asDouble());
    /////////////////////////////////////////////////////////////////////////

    dg::Grid2d grid( 0, p.lx, 0, p.ly, p.n, p.Nx, p.Ny, p.bc_x, p.bc_y);
    //create RHS 
    toefl::Explicit<dg::CartesianGrid2d, dg::DMatrix, dg::DVec > test( grid, p); 
    toefl::Implicit<dg::CartesianGrid2d, dg::DMatrix, dg::DVec> diffusion( grid, p.nu);
    //////////////////create initial vector///////////////////////////////////////
    dg::Gaussian g( p.posX*p.lx, p.posY*p.ly, p.sigma, p.sigma, p.amp); //gaussian width is in absolute values
    std::vector<dg::DVec> y0(2, dg::evaluate( g, grid)), y1(y0); // n_e' = gaussian
    dg::blas2::symv( test.gamma(), y0[0], y0[1]); // n_e = \Gamma_i n_i -> n_i = ( 1+alphaDelta) n_e' + 1
    {
        dg::DVec v2d = dg::create::inv_weights(grid);
        dg::blas2::symv( v2d, y0[1], y0[1]);
    }
    if( p.equations == "gravity_local" || p.equations == "gravity_global" || p.equations == "drift_global"){
        y0[1] = dg::evaluate( dg::zero, grid);
    }
    //////////////////////////////////////////////////////////////////////


    //dg::AB< k, std::vector<dg::DVec> > ab( y0);
    dg::Karniadakis< std::vector<dg::DVec> > ab( y0, y0[0].size(), p.eps_time);

    dg::DVec dvisual( grid.size(), 0.);
    dg::HVec hvisual( grid.size(), 0.), visual(hvisual);
    dg::IHMatrix equi = dg::create::backscatter( grid);
    draw::ColorMapRedBlueExt colors( 1.);
    //create timer
    dg::Timer t;
    double time = 0;
    ab.init( test, diffusion, y0, p.dt);
    const double mass0 = test.mass(), mass_blob0 = mass0 - grid.lx()*grid.ly();
    double E0 = test.energy(), energy0 = E0, E1 = 0, diff = 0;
    std::cout << "Begin computation \n";
    std::cout << std::scientific << std::setprecision( 2);
    unsigned step = 0;
    while ( !glfwWindowShouldClose( w ))
    {
        //transform field to an equidistant grid
        dvisual = y0[0];

        dg::blas1::transfer( dvisual, hvisual);
        dg::blas2::gemv( equi, hvisual, visual);
        //compute the color scale
        colors.scale() =  (float)thrust::reduce( visual.begin(), visual.end(), 0., dg::AbsMax<double>() );
        //draw ions
        title << std::setprecision(2) << std::scientific;
        title <<"ne / "<<colors.scale()<<"\t";
        render.renderQuad( visual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);

        //transform phi
        dvisual = test.potential()[0];
        dg::blas2::gemv( test.laplacianM(), dvisual, y1[1]);
        dg::blas1::transfer( y1[1], hvisual);
        dg::blas2::gemv( equi, hvisual, visual);
        //compute the color scale
        colors.scale() =  (float)thrust::reduce( visual.begin(), visual.end(), 0., dg::AbsMax<double>() );
        //draw phi and swap buffers
        title <<"omega / "<<colors.scale()<<"\t";
        title << std::fixed; 
        title << " &&   time = "<<time;
        render.renderQuad( visual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        glfwSetWindowTitle(w,title.str().c_str());
        title.str("");
        glfwPollEvents();
        glfwSwapBuffers( w);

        //step 
#ifdef DG_BENCHMARK
        t.tic();
#endif//DG_BENCHMARK
        for( unsigned i=0; i<p.itstp; i++)
        {
            step++;
            {
                std::cout << "(m_tot-m_0)/m_0: "<< (test.mass()-mass0)/mass_blob0<<"\t";
                E0 = E1;
                E1 = test.energy();
                diff = (E1 - E0)/p.dt;
                double diss = test.energy_diffusion( );
                std::cout << "(E_tot-E_0)/E_0: "<< (E1-energy0)/energy0<<"\t";
                std::cout << "Accuracy: "<< 2.*(diff-diss)/(diff+diss)<<"\n";

            }
            try{ ab( test, diffusion, y0);}
            catch( dg::Fail& fail) { 
                std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
                std::cerr << "Does Simulation respect CFL condition?\n";
                glfwSetWindowShouldClose( w, GL_TRUE);
                break;
            }
        }
        time += (double)p.itstp*p.dt;
#ifdef DG_BENCHMARK
        t.toc();
        std::cout << "\n\t Step "<<step;
        std::cout << "\n\t Average time for one step: "<<t.diff()/(double)p.itstp<<"s\n\n";
#endif//DG_BENCHMARK
    }
    glfwTerminate();
    ////////////////////////////////////////////////////////////////////

    return 0;

}
